#include "hip/hip_runtime.h"
/* -*- C++ -*- */
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <hipblas.h>

/*
  matrix multiply A = B*C on CUDA
*/

#ifdef _DOUBLE
#define real double
#endif
#ifdef _SINGLE
#define real float
#endif

#include "time_cpu_struct.h"
#include "time_gpu_struct.h"
struct SGpuTime gputimeH2D_1;
struct SGpuTime gputimeH2D_2;
struct SGpuTime gputimeKernel;
struct SGpuTime gputimeD2H;

#define TIME_INITIALIZE();							\
  TimeInitialize_GPU(&gputimeH2D_1,"gputimeH2D_1");\
  TimeInitialize_GPU(&gputimeH2D_2,"gputimeH2D_2");\
  TimeInitialize_GPU(&gputimeKernel,"gputimeKernel");\
  TimeInitialize_GPU(&gputimeD2H,"gputimeD2H");
#define DO_H2D_1(o); TimeBegin_GPU(&gputimeH2D_1); ret=o; TimeEnd1_GPU(&gputimeH2D_1);
#define DO_H2D_2(o); TimeBegin_GPU(&gputimeH2D_2); ret=o; TimeEnd1_GPU(&gputimeH2D_2);
#define DO_KERNEL3(o,n); TimeBegin_GPU(&gputimeKernel); ret=o; TimeEnd2_GPU(&gputimeKernel, n);
#define DO_KERNEL2(o,n); TimeBegin_GPU(&gputimeKernel); ret=o; TimeEnd2_GPU(&gputimeKernel, 1.0e-9*2.0*nelem);
#define DO_KERNEL1(o); TimeBegin_GPU(&gputimeKernel); ret=o; TimeEnd1_GPU(&gputimeKernel);
#define DO_D2H(o); TimeBegin_GPU(&gputimeD2H); ret=o; TimeEnd1_GPU(&gputimeD2H);

int SIZE = 16;
int RANDSEED = 0;
real *g_A, *g_B, *g_C;
real *g_dbg;
int KERNEL = 0;
real *d_A, *d_B, *d_C;
real *d_dbg;
hipblasStatus_t stat;
hipblasHandle_t handle;
int BLOCKS=256, THREADS=32;
int LOOPS=1;
int SMLIMIT=0;

int checkArgs(int argc, char** argv)
{
  int i;
  for(i=1; i<argc; i++){
	if(strcmp(argv[i], "-size")==0){
	  SIZE = atoi(argv[++i]);
	}
	if(strcmp(argv[i], "-kernel")==0){
	  KERNEL = atoi(argv[++i]);
	}
	if(strcmp(argv[i], "-blocks")==0){
	  BLOCKS = atoi(argv[++i]);
	  printf(" %d block(s)\n", BLOCKS);
	}
	if(strcmp(argv[i], "-threads")==0){
	  THREADS = atoi(argv[++i]);
	  printf(" %d thread(s)\n", THREADS);
	}
	if(strcmp(argv[i], "-loops")==0){
	  LOOPS = atoi(argv[++i]);
	  printf(" %d loop(s)\n", LOOPS);
	}
	if(strcmp(argv[i], "-smlimit")==0){
	  SMLIMIT = atoi(argv[++i]);
	  printf(" SMLIMIT %d\n", SMLIMIT);
	}
  }
  return 0;
}

real frand()
{
  //return (real)(rand()%1000) / 1000.0;
  return (real)((double)(rand()%10) / 5.0);
}

int gpu_h2d(void *d, void *h, int elemsize, int count)
{
  hipMemcpy(d,h,elemsize*count,hipMemcpyHostToDevice);
  return 0;
}

int gpu_d2h(void *h, void *d, int elemsize, int count)
{
  hipMemcpy(h,d,elemsize*count,hipMemcpyDeviceToHost);
  return 0;
}

// 逐次
__global__ void gpu_kernel_1(int size, real *_a, real *_b, real *_c)
{
  int i, j, k;
  for(j=0; j<size; j++){
	for(i=0; i<size; i++){
	  real sum = 0.0f;
	  for(k=0; k<size; k++){
		sum += _b[j*size+k] * _c[k*size+i];
	  }
	  _a[j*size+i] = sum;
	}
  }
}

// 行単位並列化
__global__ void gpu_kernel_2(int size, real *_a, real *_b, real *_c)
{
  int i, j, k;
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  int jBegin = id;
  int jEnd = size;
  int jStep = gridDim.x*blockDim.x;
  for(j=jBegin; j<jEnd; j+=jStep){
	for(i=0; i<size; i++){
	  real tmp = 0.0;
	  for(k=0; k<size; k++){
		tmp += _b[j*size+k] * _c[k*size+i];
	  }
	  _a[j*size+i] = tmp;
	}
  }
}

// 行単位並列化：blockとthreadを明確に分離
__global__ void gpu_kernel_3(int size, real *_a, real *_b, real *_c)
{
  __shared__ real sm[32];
  int i, j, k;
  for(j=blockIdx.x; j<size; j+=gridDim.x){
	__syncthreads();
	for(i=0; i<size; i++){
	  sm[threadIdx.x] = 0.0;
	  for(k=threadIdx.x; k<size; k+=blockDim.x){
		sm[threadIdx.x] += _b[j*size+k] * _c[k*size+i];
	  }
	  __syncthreads();
	  if(threadIdx.x==0){
		for(k=0; k<32; k++){
		  _a[j*size+i] += sm[k];
		}
	  }
	  __syncthreads();
	}
  }
}

// SM blocking
// SXthreads, Xblocks
__global__ void gpu_kernel_4(int size, real *_a, real *_b, real *_c, real *_dbg)
{
  const int SX = 32;
  __shared__ real smA[SX*SX];
  __shared__ real smB[SX*SX];
  __shared__ real smC[SX*SX];
  int x, y, z;
  int i, j, k, l;
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bs = SX;

  for(y=bid*bs; y<size; y+=bs*gridDim.x){
	for(x=0; x<size; x+=bs){
	  for(i=0; i<SX; i++){
		smA[i*SX+tid] = 0.0;
	  }
	  for(z=0; z<size/SX; z++){
		for(i=0; i<SX; i++){
		  smB[i*SX+tid] = _b[(y+i)*size +z*SX+tid];
		  smC[i*SX+tid] = _c[(i+z*SX)*size +x+tid];
		}
		__syncthreads();
		{
		  for(i=0; i<SX; i++){
			//for(j=0; j<SX; j++)
			j = tid;
			{
			  real tmp = 0.0;
			  for(k=0; k<SX; k++){
				tmp += smB[i*SX+k] * smC[k*SX+j];
			  }
			  smA[i*SX+j] += tmp;
			}
		  }
		}
		__syncthreads();
	  }
	  for(i=0; i<SX; i++){
		_a[(y+i)*size+x+tid] = smA[i*SX+tid];
	  }
	  __syncthreads();
	}
  }
}
// SM blocking
template <int SX>
__global__ void gpu_kernel_5(int size, real * __restrict__ _a, real * __restrict__  _b, real * __restrict__  _c)
{
  int x, y, z;
  int k, l;
  int tidX = threadIdx.x;
  int tidY = threadIdx.y;
  int bid = blockIdx.x;
  int bs1 = SX;
  int bs2 = bs1*gridDim.x;

  for(y=bid*bs1; y<size; y+=bs2){
	for(x=0; x<size; x+=bs1){
#if 1
	  real tmp = 0.0;
	  __shared__ real smB[SX*SX];
	  __shared__ real smC[SX*SX];
	  for(z=0; z<size; z+=SX){
		smB[tidY*SX +tidX] = _b[(y+tidY)*size +z+tidX];
		smC[tidY*SX +tidX] = _c[(z+tidY)*size +x+tidX];
		__syncthreads();
#pragma unroll
		for(k=0; k<SX; k++){
		  tmp += smB[tidY*SX+k] * smC[k*SX+tidX];
		}
		__syncthreads();
	  }
#else
	  real tmp = 0.0;
	  __shared__ real smB[SX][SX];
	  __shared__ real smC[SX][SX];
	  for(z=0; z<size; z+=SX){
		smB[tidY][tidX] = _b[(y+tidY)*size +z+tidX];
		smC[tidY][tidX] = _c[(z+tidY)*size +x+tidX];
		__syncthreads();
#pragma unroll
		for(k=0; k<SX; k++){
		  tmp += smB[tidY][k] * smC[k][tidX];
		}
		__syncthreads();
	  }
#endif
	  _a[(y+tidY)*size +x+tidX] = tmp;
	}
  }
}

int gpu_kernel(int kernel, int size, real *_a, real *_b, real *_c, real *_dbg)
{
  switch(kernel){
  default:
  case 0:
	printf("kernel %d is undefined\n", kernel);
	return -1;
	break;
  case 1:
	gpu_kernel_1<<<1,1>>>(size,_a,_b,_c);
	break;
  case 2:
	//gpu_kernel_2<<<240,256>>>(size,_a,_b,_c); // (448/32)*10=240
	gpu_kernel_2<<<624,256>>>(size,_a,_b,_c); // (2496/32)*8=624
	break;
  case 3:
	gpu_kernel_3<<<624,32>>>(size,_a,_b,_c);
	break;
  case 4:
	gpu_kernel_4<<<624,32>>>(size,_a,_b,_c, _dbg);
	break;
  case 5:
	//gpu_kernel_5<<<624,dim3(32,32,1)>>>(size,_a,_b,_c);
	{
	  switch(THREADS){
	  case 32:	gpu_kernel_5<32><<<BLOCKS,dim3(32,32,1)>>>(size,_a,_b,_c);	break;
	  case 24:	gpu_kernel_5<24><<<BLOCKS,dim3(24,24,1)>>>(size,_a,_b,_c);	break;
	  case 16:	gpu_kernel_5<16><<<BLOCKS,dim3(16,16,1)>>>(size,_a,_b,_c);	break;
	  case  8:	gpu_kernel_5< 8><<<BLOCKS,dim3( 8, 8,1)>>>(size,_a,_b,_c);	break;
	  case  4:	gpu_kernel_5< 4><<<BLOCKS,dim3( 4, 4,1)>>>(size,_a,_b,_c);	break;
	  default:	gpu_kernel_5< 2><<<BLOCKS,dim3( 2, 2,1)>>>(size,_a,_b,_c);	break;
	  }
	}
	break;
  case 99:
	{
	  real alpha = 1.0;
	  real beta = 0.0;
#ifdef _DOUBLE
	  hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, size, size, size, &alpha, d_B, SIZE, d_C, size, &beta, d_A, size);
#else
	  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, size, size, size, &alpha, d_B, SIZE, d_C, size, &beta, d_A, size);
#endif
	}
	break;
  }
  return 0;
}

// ******** ******** ******** ********
// main
// ******** ******** ******** ********
int main(int argc, char** argv)
{
  int i;
  if(checkArgs(argc,argv))return -1;

  printf("size %d\n", SIZE);
  srand(RANDSEED);

  TIME_INITIALIZE();

  //printf("initialize...");
  int t,g;

  g_A = (real*)malloc(sizeof(real)*SIZE*SIZE);
  g_B = (real*)malloc(sizeof(real)*SIZE*SIZE);
  g_C = (real*)malloc(sizeof(real)*SIZE*SIZE);
  hipMalloc((void**)&d_A, sizeof(real)*SIZE*SIZE);
  hipMalloc((void**)&d_B, sizeof(real)*SIZE*SIZE);
  hipMalloc((void**)&d_C, sizeof(real)*SIZE*SIZE);
  g_dbg = (real*)malloc(sizeof(real)*1000);
  hipMalloc((void**)&d_dbg, sizeof(real)*1000);
  for(i=0;i<1000;i++)g_dbg[i] = 0.0;

  for(g=0; g<SIZE; g++){
	for(t=0; t<SIZE; t++){
	  g_A[g*SIZE+t] = 0.0;
	  g_B[g*SIZE+t] = frand();
	  g_C[g*SIZE+t] = frand();
	}
  }
  /*
  printf("done\n");
  printf("====\n");
  for(g=0; g<SIZE; g++){
	for(t=0; t<SIZE; t++){
	  printf(" %.2f", g_B[g*SIZE+t]);
	}
	printf("\n");
  }
  printf("====\n");
  */
  {
	int i, j;
	FILE *F;
	F = fopen("B.txt", "w");
	for(i=0; i<SIZE; i++){
	  for(j=0; j<SIZE; j++){
		fprintf(F, " %.2f", g_B[i*SIZE+j]);
	  }
	  fprintf(F, "\n");
	}
	fclose(F);
  }
  {
	int i, j;
	FILE *F;
	F = fopen("C.txt", "w");
	for(i=0; i<SIZE; i++){
	  for(j=0; j<SIZE; j++){
		fprintf(F, " %.2f", g_C[i*SIZE+j]);
	  }
	  fprintf(F, "\n");
	}
	fclose(F);
  }

  struct timeval tBegin, tEnd;
  struct timezone tz;
  real dSec;
  real dBegin, dEnd;
  int ret;
  if(KERNEL!=99){
	gpu_h2d(d_A, g_A, sizeof(real), SIZE*SIZE);
	gpu_h2d(d_B, g_B, sizeof(real), SIZE*SIZE);
	gpu_h2d(d_C, g_C, sizeof(real), SIZE*SIZE);
	gpu_h2d(d_dbg, g_dbg, sizeof(real), 1000);
  }
  if(KERNEL==99){
	stat = hipblasCreate(&handle);
	if(stat != HIPBLAS_STATUS_SUCCESS){
	  printf("CUBLAS initialization failed\n");
	}
	stat = hipblasSetMatrix(SIZE,SIZE,sizeof(real),g_A,SIZE,d_A,SIZE);
	if(stat != HIPBLAS_STATUS_SUCCESS){
	  printf("CUBLAS SetMatrix A failed\n");
	  return -1;
	}
	stat = hipblasSetMatrix(SIZE,SIZE,sizeof(real),g_B,SIZE,d_B,SIZE);
	if(stat != HIPBLAS_STATUS_SUCCESS){
	  printf("CUBLAS SetMatrix B failed\n");
	  return -1;
	}
	stat = hipblasSetMatrix(SIZE,SIZE,sizeof(real),g_C,SIZE,d_C,SIZE);
	if(stat != HIPBLAS_STATUS_SUCCESS){
	  printf("CUBLAS SetMatrix C failed\n");
	  return -1;
	}
	if(SMLIMIT>0){
	  printf("cublasSetSmCountTarget %d\n", SMLIMIT);
	  stat = cublasSetSmCountTarget(handle, SMLIMIT);
	  if(stat != HIPBLAS_STATUS_SUCCESS){
		printf("cublasSetSmCountTarget failed\n");
		return -1;
	  }
	}

  }
  gettimeofday(&tBegin, &tz);
  for(i=0;i<LOOPS;i++){DO_KERNEL3(gpu_kernel(KERNEL, SIZE, d_A, d_B, d_C, d_dbg), 1.0e-9*SIZE*SIZE*SIZE*2);}
  if(ret!=0)return -1;
  gettimeofday(&tEnd, &tz);
  if(KERNEL!=99){
	gpu_d2h(g_A, d_A, sizeof(real), SIZE*SIZE);
	gpu_d2h(g_dbg, d_dbg, sizeof(real), 1000);
  }else{
	stat = hipblasGetMatrix(SIZE,SIZE,sizeof(real),d_A,SIZE,g_A,SIZE);
	if(stat != HIPBLAS_STATUS_SUCCESS){
	  printf("CUBLAS GetMatrix failed\n");
	}
  }
  dBegin= tBegin.tv_sec + (double)tBegin.tv_usec*1.0e-6;
  dEnd= tEnd.tv_sec + (double)tEnd.tv_usec*1.0e-6;
  dSec= dEnd - dBegin;
  printf("performance: size %d^2, %f sec, %f Mflops\n", SIZE, dSec, (double)(LOOPS*2.0*(double)SIZE*(double)SIZE*(double)SIZE)/dSec/1000.0/1000.0);
  TimePrintf_GPU(&gputimeKernel);

  printf("random sampling:");
  for(i=0; i<10; i++){
	int n = rand()%(SIZE*SIZE);
	printf(" %.2f", g_A[n]);
  }
  printf("\n");

  {
	int i, j;
	FILE *F;
	F = fopen("dbg.txt", "w");
	for(i=0; i<1000; i++){
	  fprintf(F, " %.2f", g_dbg[i]);
	  if((i+1)%8==0)fprintf(F, "\n");
	}
	fprintf(F, "\n");
	fclose(F);
  }

  {
	int i, j;
	FILE *F;
	char filename[16];
	snprintf(filename, 16, "A_cuda_k%d.txt", KERNEL);
	F = fopen(filename, "w");
	for(i=0; i<SIZE; i++){
	  for(j=0; j<SIZE; j++){
		fprintf(F, " %.2f", g_A[i*SIZE+j]);
	  }
	  fprintf(F, "\n");
	}
	fclose(F);
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  if(KERNEL!=99){
  }else{
	hipblasDestroy(handle);
  }
  free(g_A);
  free(g_B);
  free(g_C);

  return 0;
}
