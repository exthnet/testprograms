#include "hip/hip_runtime.h"
/* -*- C++ -*- */
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C"
int gpu_h2d(void *d, void *h, int elemsize, int count)
{
  hipMemcpy(d,h,elemsize*count,hipMemcpyHostToDevice);
  return 0;
}

extern "C"
int gpu_d2h(void *h, void *d, int elemsize, int count)
{
  hipMemcpy(h,d,elemsize*count,hipMemcpyDeviceToHost);
  return 0;
}

// 逐次
__global__ void gpu_kernel_1(int size, double *_a, double *_b, double *_c)
{
  int i, j, k;
  for(j=0; j<size; j++){
	for(i=0; i<size; i++){
	  double sum = 0.0f;
	  for(k=0; k<size; k++){
		sum += _b[j*size+k] * _c[k*size+i];
	  }
	  _a[j*size+i] = sum;
	}
  }
}

// 行単位並列化
__global__ void gpu_kernel_2(int size, double *_a, double *_b, double *_c)
{
  int i, j, k;
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  int jBegin = id;
  int jEnd = size;
  int jStep = gridDim.x*blockDim.x;
  for(j=jBegin; j<jEnd; j+=jStep){
	for(i=0; i<size; i++){
	  double sum = 0.0f;
	  for(k=0; k<size; k++){
		sum += _b[j*size+k] * _c[k*size+i];
	  }
	  _a[j*size+i] = sum;
	}
  }
}

// SM blocking
// 32threads, Xblocks
__global__ void gpu_kernel_3(int size, double *_a, double *_b, double *_c)
{
  __shared__ double smA[32*32];
  __shared__ double smB[32*32];
  __shared__ double smC[32*32];
  int x, y;
  int i, j, k, l;
  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int bs = 32;
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  int jBegin = id;
  int jEnd = size;
  int jStep = gridDim.x*blockDim.x;
  for(y=bid*bs; y<size; y+=bs){
	for(x=0; x<size; x+=bs){

	  for(i=0; i<32; i++){
		smA[i*32+tid] = 0.0;
		smB[i*32+tid] = _b[(y+i)*32+x+tid];
		smC[tid*32+i] = _c[(y+i)*32+x+tid];
	  }
	  if(tid==0){
		for(i=0; i<32; i++){
		  for(j=0; j<32; j++){
			for(k=0; k<32; k++){
			  smA[i*32+j] += smB[i*32+k] * smC[i*32+k];
			}
		  }
		}
	  }
	  for(i=0; i<32; i++){
		_a[(i+1)*32+x+tid] = smA[i*32+tid];
	  }
	}
  }
}

extern "C"
int gpu_kernel(int kernel, int size, double *_a, double *_b, double *_c)
{
  switch(kernel){
  case 0:
	DO_KERNEL(gpu_kernel_1<<<1,1>>>(size,_a,_b,_c));
	break;
  case 1:
	//DO_KERNEL(gpu_kernel_2<<<240,256>>>(size,_a,_b,_c)); // (448/32)*10
	DO_KERNEL(gpu_kernel_2<<<624,256>>>(size,_a,_b,_c)); // (2496/32)*8
	break;
  case 2:
	DO_KERNEL(gpu_kernel_3<<<624,32>>>(size,_a,_b,_c));
	break;
  default:
	printf("kernel %d is undefined\n", kernel);
	break;
  }
  return 0;
}
