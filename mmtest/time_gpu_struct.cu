// -*- C++ -*-
#include "time_gpu_struct.h"

// timeGetTime
extern "C"
int TimeInitialize_GPU(struct SGpuTime *t, const char *str){
  t->dSec = 0.0;
  t->dMinSec = 9999999.0;
  t->dMaxSec = 0.0;
  t->dFlops = 0.0;
  t->nFlops = 0;
  t->nActive = 0;
  t->nCount = 0;
  strcpy(t->name, str);
  return 0;
}
int TimeBegin_GPU(struct SGpuTime *t){
  if(t->nActive == 0){
    gettimeofday(&t->tBegin, &t->tz);
    t->nActive = 1;
  }else{
    printf("%s *** already activated\n", t->name);
    return 1;
  }
  return 0;
}
int TimeEnd1_GPU(struct SGpuTime *t){
  if(t->nActive == 1){
    double sec;
    hipError_t _err = hipGetLastError();
    if(hipSuccess!=_err){
      printf("%s error: time_gpu_end %s\n", t->name, hipGetErrorString(_err));
      return 1;
    }
    _err=hipDeviceSynchronize();
    if(hipSuccess!=_err){
      printf("%s error: time_gpu_end %s\n", t->name, hipGetErrorString(_err));
      return 1;
    }
    double dBegin, dEnd;
    gettimeofday(&t->tEnd, &t->tz);
    dBegin= t->tBegin.tv_sec + (double)t->tBegin.tv_usec*1.0e-6;
    dEnd= t->tEnd.tv_sec + (double)t->tEnd.tv_usec*1.0e-6;
    sec = dEnd - dBegin;
    t->dSec+= sec;
    if(sec > t->dMaxSec)t->dMaxSec = sec;
    if(sec < t->dMinSec)t->dMinSec = sec;
	t->nCount++;
    t->nFlops = 0;
    t->nActive = 0;
  }else{
    printf("%s *** not activated yet\n", t->name);
    return 2;
  }
  return 0;
}
int TimeEnd2_GPU(struct SGpuTime *t, double flop){
  if(t->nActive == 1){
    double sec;
    hipError_t _err = hipGetLastError();
    if(hipSuccess!=_err){
      printf("%s error: time_gpu_end %s\n", t->name, hipGetErrorString(_err));
      return 1;
    }
    _err=hipDeviceSynchronize();
    if(hipSuccess!=_err){
      printf("%s error: time_gpu_end %s\n", t->name, hipGetErrorString(_err));
      return 1;
    }
    double dBegin, dEnd;
    gettimeofday(&t->tEnd, &t->tz);
    dBegin= t->tBegin.tv_sec + (double)t->tBegin.tv_usec*1.0e-6;
    dEnd= t->tEnd.tv_sec + (double)t->tEnd.tv_usec*1.0e-6;
    sec = dEnd - dBegin;
    t->dSec+= sec;
    if(sec > t->dMaxSec)t->dMaxSec = sec;
    if(sec < t->dMinSec)t->dMinSec = sec;
	t->nCount++;
    double flops = flop/sec;
	if(t->nCount==1){
	  t->dFlops = flops;
	}else{
	  t->dFlops = (t->dFlops+flops)/2.0;
	}
    t->nFlops = 1;
    t->nActive = 0;
    return 0;
  }else{
    printf("%s *** not activated yet\n", t->name);
    return 2;
  }
}
int TimePrintf_GPU(struct SGpuTime *t){
  if(t->nActive == 0){
    if(t->nFlops==0){
      printf("%s %f sec ( %f - %f sec ) / %d times\n", t->name, t->dSec, t->dMinSec, t->dMaxSec, t->nCount);
    }else{
      printf("%s %f sec, %f Gflops ( %f - %f sec ) / %d times\n", t->name, t->dSec, t->dFlops, t->dMinSec, t->dMaxSec, t->nCount);
    }
    return 0;
  }else{
    printf("%s %f sec ( %f - %f sec )\n", t->name, 0.0, 0.0, 0.0);
    //printf("%s *** still activated\n", t->name);
    return 2;
  }
}
