// -*- c++ -*-
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
//#include <cuda_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define frand() (double)(rand()%100)/10.0

__global__ void gpukernel
(int N, double *A, double *B, double *C){
  int i, j, k, ibegin, istep, jbegin, jstep;
  ibegin = blockIdx.y*blockDim.y + threadIdx.y;
  istep = blockDim.y * gridDim.y;
  jbegin = blockIdx.x*blockDim.x + threadIdx.x;
  jstep = blockDim.x * gridDim.x;
  for(i=ibegin; i<N; i+=istep){
	for(j=jbegin; j<N; j+=jstep){
	  for(k=0; k<N; k++){
		C[i*N+j] += A[i*N+k] * B[k*N+j];
	  }
	}
  }
}

double *A, *B, *C;
double *d_A, *d_B, *d_C; // device memory
int N = 100;
int blk = 10;
int main(int argc, char **argv){
  if(argc>1){
	N = atoi(argv[1]);
  }
  if(argc>2){
	blk = atoi(argv[2]);
  }
  printf("N = %d, blk = %d\n", N, blk);

  hipError_t retDev;
  hipDevice_t dev, dev2;
  CUdevResource devRsc, devRsc2, devRsc3;
  unsigned int nGroups, nMin=16;
  CUdevResourceDesc devRscDesc, devRscDesc2;
  CUgreenCtx phCtx;
  retDev = hipInit(0);
  if(retDev!=hipSuccess){printf("hipInit failed\n"); return -1;}
  retDev = hipDeviceGet(&dev, 0);
  if(retDev!=hipSuccess){printf("hipDeviceGet failed\n"); return -1;}
  retDev = cuDeviceGetDevResource(dev, &devRsc, CU_DEV_RESOURCE_TYPE_SM);
  if(retDev!=hipSuccess){printf("cuDeviceGetResource failed\n"); return -1;}
  retDev = cuDevSmResourceSplitByCount(&devRsc2, &nGroups, &devRsc, &devRsc3, 0, nMin);
  //retDev = cuDevSmResourceSplitByCount(NULL, &nGroups, &devRsc, NULL, 0, nMin);
  if(retDev!=hipSuccess){printf("hipDevSmResourceSplitByCount failed\n");
	switch(retDev){
	case hipSuccess:
	  printf("CUDA_SUCCESS\n"); break;
	case hipErrorDeinitialized:
	  printf("CUDA_ERROR_DEINITIALIZED\n"); break;
	case hipErrorNotInitialized:
	  printf("CUDA_ERROR_NOT_INITIALIZED\n"); break;
	case hipErrorInvalidDevice:
	  printf("CUDA_ERROR_INVALID_DEVICE\n"); break;
	case hipErrorInvalidValue:
	  printf("CUDA_ERROR_INVALID_VALUE\n"); break;
	case CUDA_ERROR_INVALID_RESOURCE_TYPE:
	  printf("CUDA_ERROR_INVALID_RESOURCE_TYPE\n"); break;
	case CUDA_ERROR_INVALID_RESOURCE_CONFIGURATION:
	  printf("CUDA_ERROR_INVALID_RESOURCE_CONFIGURATION\n"); break;
	}
  }
  retDev = cuDevResourceGenerateDesc(&devRscDesc, &devRsc2, 1);
  if(retDev!=hipSuccess){printf("cuDeviceGetResource failed\n");}
  retDev = cuGreenCtxCreate(&phCtx, devRscDesc, dev2, CU_GREEN_CTX_DEFAULT_STREAM);
  if(retDev!=hipSuccess){printf("hipGreenCtxCreate failed\n");}

  int x, y;
  A = (double*)malloc(sizeof(double)*N*N);
  B = (double*)malloc(sizeof(double)*N*N);
  C = (double*)malloc(sizeof(double)*N*N);
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      A[y*N+x] = frand();
      B[y*N+x] = frand();
      C[y*N+x] = 0.0;
    }
  }
  hipMalloc((void**)&d_A, sizeof(double)*N*N);
  hipMalloc((void**)&d_B, sizeof(double)*N*N);
  hipMalloc((void**)&d_C, sizeof(double)*N*N);
  hipMemcpy(d_A, A, sizeof(double)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, sizeof(double)*N*N, hipMemcpyHostToDevice);

  double d1, d2;
  hipDeviceSynchronize();
  d1 = omp_get_wtime();
  gpukernel<<<dim3(blk,blk),dim3(8,8)>>>(N, d_A, d_B, d_C);
  hipDeviceSynchronize();
  d2 = omp_get_wtime();
  printf("gpukernel %f sec\n", d2-d1);

  hipMemcpy(C, d_C, sizeof(double)*N*N, hipMemcpyDeviceToHost);
  hipFree(d_A); hipFree(d_B); hipFree(d_C);

  // check result
  {
	int i;
	for(i=0; i<10; i++)printf(" %f", A[i]); printf("\n");
	for(i=0; i<10; i++)printf(" %f", B[i]); printf("\n");
	for(i=0; i<10; i++)printf(" %f", C[i]); printf("\n");
	double sum=0.0;
	for(y=0; y<N; y++){
	  for(x=0; x<N; x++){
		sum += C[y*N+x];
	  }
	}
	printf("sum %f\n", sum);
  }

  /*
  // A
  printf("A\n");
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      printf(" %.2f", A[y*N+x]);
    }
	printf("\n");
  }
  // B
  printf("B\n");
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      printf(" %.2f", B[y*N+x]);
    }
	printf("\n");
  }
  // C
  printf("C\n");
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      printf(" %.2f", C[y*N+x]);
    }
	printf("\n");
  }
  */
  free(A); free(B); free(C);
  return 0;
}
