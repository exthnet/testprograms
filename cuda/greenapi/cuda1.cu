// -*- c++ -*-
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#define frand() (double)(rand()%100)/10.0

__global__ void gpukernel
(int N, double *A, double *B, double *C){
  int i, j, k, ibegin, istep, jbegin, jstep;
  ibegin = blockIdx.y*blockDim.y + threadIdx.y;
  istep = blockDim.y * gridDim.y;
  jbegin = blockIdx.x*blockDim.x + threadIdx.x;
  jstep = blockDim.x * gridDim.x;
  for(i=ibegin; i<N; i+=istep){
	for(j=jbegin; j<N; j+=jstep){
	  for(k=0; k<N; k++){
		C[i*N+j] += A[i*N+k] * B[k*N+j];
	  }
	}
  }
}

double *A, *B, *C;
double *d_A, *d_B, *d_C; // device memory
int N = 100;
int blk = 10;
int main(int argc, char **argv){
  if(argc>1){
	N = atoi(argv[1]);
  }
  if(argc>2){
	blk = atoi(argv[2]);
  }
  printf("N = %d, blk = %d\n", N, blk);
  int x, y;
  A = (double*)malloc(sizeof(double)*N*N);
  B = (double*)malloc(sizeof(double)*N*N);
  C = (double*)malloc(sizeof(double)*N*N);
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      A[y*N+x] = frand();
      B[y*N+x] = frand();
      C[y*N+x] = 0.0;
    }
  }
  hipMalloc((void**)&d_A, sizeof(double)*N*N);
  hipMalloc((void**)&d_B, sizeof(double)*N*N);
  hipMalloc((void**)&d_C, sizeof(double)*N*N);
  hipMemcpy(d_A, A, sizeof(double)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(double)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, sizeof(double)*N*N, hipMemcpyHostToDevice);

  double d1, d2;
  hipDeviceSynchronize();
  d1 = omp_get_wtime();
  gpukernel<<<dim3(blk,blk),dim3(8,8)>>>(N, d_A, d_B, d_C);
  hipDeviceSynchronize();
  d2 = omp_get_wtime();
  printf("gpukernel %f sec\n", d2-d1);

  hipMemcpy(C, d_C, sizeof(double)*N*N, hipMemcpyDeviceToHost);
  hipFree(d_A); hipFree(d_B); hipFree(d_C);

  // check result
  {
	int i;
	for(i=0; i<10; i++)printf(" %f", A[i]); printf("\n");
	for(i=0; i<10; i++)printf(" %f", B[i]); printf("\n");
	for(i=0; i<10; i++)printf(" %f", C[i]); printf("\n");
	double sum=0.0;
	for(y=0; y<N; y++){
	  for(x=0; x<N; x++){
		sum += C[y*N+x];
	  }
	}
	printf("sum %f\n", sum);
  }

  /*
  // A
  printf("A\n");
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      printf(" %.2f", A[y*N+x]);
    }
	printf("\n");
  }
  // B
  printf("B\n");
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      printf(" %.2f", B[y*N+x]);
    }
	printf("\n");
  }
  // C
  printf("C\n");
  for(y=0; y<N; y++){
    for(x=0; x<N; x++){
      printf(" %.2f", C[y*N+x]);
    }
	printf("\n");
  }
  */
  free(A); free(B); free(C);
  return 0;
}
