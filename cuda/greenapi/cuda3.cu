#include <hip/hip_runtime.h>

#include <cstdio>
#include <cassert>
#include <cstdint>

#define CUDA_RT(call)                                                   \
    do {                                                                \
        hipError_t _err = (call);                                      \
        if ( hipSuccess != _err ) {                                    \
            fprintf(stderr, "CUDA error in file '%s' at line %i: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(_err));      \
            return _err;                                                \
        } } while (0)


#define CUDA_DRV(call)                                                  \
    do {                                                                \
        hipError_t _status = (call);                                      \
        if ( hipSuccess != _status) {                                 \
            fprintf(stderr, "CUDA error in file '%s' at line %i: %i\n", \
                    __FILE__, __LINE__, _status);                       \
            return _status;                                             \
        } } while (0)

__device__ int temp_result;

extern "C" __global__ void timewaster(const int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    volatile float result = 0;
    for (int i = 0; i < num_iterations; i++) {
        result += sqrtf(i + idx) + sinf(idx * i);
    }
    if (idx == 0) {
        temp_result = (int)result;
    }
}

__global__ void kernel()
{
    temp_result += threadIdx.x * blockIdx.x;
}

int main()
{
    CUgreenCtx gctx[2];
    CUdevResourceDesc desc[2];
    CUdevResource input;
    CUdevResource resources[2];
    hipStream_t streamA;
    hipStream_t streamB;

    unsigned int nbGroups = 1;
    unsigned int minCount = 0;

    // Initialize device 0
    CUDA_RT(cudaInitDevice(0, 0, 0));
    // Preload
    timewaster<<<1, 512>>>(1);
    kernel<<<1, 512>>>();

    // Query input SMs
    CUDA_DRV(cuDeviceGetDevResource((hipDevice_t)0, &input, CU_DEV_RESOURCE_TYPE_SM));
    // We want 3/4 the device for our green context
    minCount = (unsigned int)((float)input.sm.smCount * 0.75f);

    // Split my resources
    CUDA_DRV(cuDevSmResourceSplitByCount(&resources[0], &nbGroups, &input, &resources[1], 0, minCount));

    // Create a descriptor/ctx for the main 3/4 partion
    CUDA_DRV(cuDevResourceGenerateDesc(&desc[0], &resources[0], 1));
    CUDA_DRV(cuGreenCtxCreate(&gctx[0], desc[0], (hipDevice_t)0, CU_GREEN_CTX_DEFAULT_STREAM));
    // ... and one for the remainder 1/4 partition
    CUDA_DRV(cuDevResourceGenerateDesc(&desc[1], &resources[1], 1));
    CUDA_DRV(cuGreenCtxCreate(&gctx[1], desc[1], (hipDevice_t)0, CU_GREEN_CTX_DEFAULT_STREAM));
    // Create streams that we will use from here on out
    CUDA_DRV(cuGreenCtxStreamCreate(&streamA, gctx[0], hipStreamNonBlocking, 0));
    CUDA_DRV(cuGreenCtxStreamCreate(&streamB, gctx[1], hipStreamNonBlocking, 0));

    timewaster<<<1000, 512, 0, (hipStream_t)streamA>>>(1000);
    kernel<<<1, 512, 0, (hipStream_t)streamB>>>();

    CUDA_RT(hipStreamSynchronize((hipStream_t)streamA));
    CUDA_RT(hipStreamSynchronize((hipStream_t)streamB));

    return (0);
}
