// -*- c++ -*-

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void gpukernel(float *B, float *A)
{
  int tid = threadIdx.x;
  float tmp = A[tid];
  if(tid>=16){
	tmp = __shfl_up_sync(0xffffffff, tmp, 2, warpSize);
  }
  if(tid<16){
	tmp = __shfl_up_sync(0xffffffff, tmp, 1, warpSize);
  }
  B[tid] = tmp;
}


int main(int argc, char **argv)
{
  int i, N;
  float *A, *B;
  float *dA, *dB;

  N = 32;
  A = (float*)malloc(sizeof(float)*N);
  B = (float*)malloc(sizeof(float)*N);

  for(i=0;i<N;i++){
	A[i] = (float)(i+1);
	B[i] = 0.0f;
  }

  hipMalloc((void**)&dA, sizeof(float)*N);
  hipMalloc((void**)&dB, sizeof(float)*N);

  printf("A\n");
  for(i=0; i<N; i++){
	printf(" %2.0f", A[i]);
  }
  printf("\n");
  printf("B (before)\n");
  for(i=0; i<N; i++){
	printf(" %2.0f", B[i]);
  }
  printf("\n");

  hipMemcpy(dA, A, sizeof(float)*N, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  gpukernel<<<1,32>>>(dB, dA);
  hipDeviceSynchronize();
  hipMemcpy(B, dB, sizeof(float)*N, hipMemcpyDeviceToHost);

  printf("B (after)\n");
  for(i=0; i<N; i++){
	printf(" %2.0f", B[i]);
  }
  printf("\n");

  hipFree(dA); hipFree(dB);
  free(A); free(B);
  return 0;
}
