// -*- c++ -*-

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <cuda.h>

__global__ void gpukernel(int N, double *C, double *A, double *B)
{
  int tid = threadIdx.x;
  double tmp = A[tid];
  //tmp = __shfl_down_sync(0xFFFFFFFF, tmp, 1, warpSize);
  //tmp = __shfl_down_sync(0x00000000, tmp, 1, warpSize);
  tmp = __shfl_up_sync(0xFFFFFFFF, tmp, 8, 16);
  /*
  if(tid<16){
	tmp = __shfl_down_sync(0x0000ffff, tmp, 1, warpSize);
  }else{
	tmp = __shfl_down_sync(0xffff0000, tmp, 2, warpSize);
  }
  */
  /*
  if(tid<16){
	tmp = __shfl_down_sync(0xffffffff, tmp, 1, warpSize);
  }else{
	tmp = __shfl_down_sync(0xffffffff, tmp, 2, warpSize);
  }
  */
  /*
  if(tid>=16){
	tmp = __shfl_down_sync(0xffffffff, tmp, 2, warpSize);
  }else{
	tmp = __shfl_down_sync(0xffffffff, tmp, 1, warpSize);
  }
  */
  /*
  if(tid>=16){
	tmp = __shfl_down_sync(0xffffffff, tmp, 2, warpSize);
  }
  if(tid<16){
	tmp = __shfl_down_sync(0xffffffff, tmp, 1, warpSize);
  }
  */
  /*
  if(tid<16){
	tmp = __shfl_down_sync(0x0000ffff, tmp, 1, warpSize);
  }
  if(tid>=16){
	tmp = __shfl_down_sync(0xffff0000, tmp, 2, warpSize);
  }
  */
  /*
  if(tid<16){
	tmp = __shfl_down_sync(0xffff0000, tmp, 1, warpSize);
  }
  if(tid>=16){
	tmp = __shfl_down_sync(0x0000ffff, tmp, 2, warpSize);
  }
  */
  C[tid] = tmp;
}


int main(int argc, char **argv)
{
  int i, N, x;
  double *A, *B, *C;
  double *dA, *dB, *dC;

  x = 32;
  N = 32;
  A = (double*)malloc(sizeof(double)*N);
  B = (double*)malloc(sizeof(double)*N);
  C = (double*)malloc(sizeof(double)*N);

  for(i=0;i<N;i++){
	C[i] = 0.0f;
	B[i] = 2.0f;
	A[i] = (double)(i+1);///(double)(N);
  }

  hipMalloc((void**)&dA, sizeof(double)*N);
  hipMalloc((void**)&dB, sizeof(double)*N);
  hipMalloc((void**)&dC, sizeof(double)*N);

  printf("A\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.0f\n", A[i]);
	}else{
	  printf(" %2.0f", A[i]);
	}
  }
  printf("\n");
  printf("B\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.0f\n", B[i]);
	}else{
	  printf(" %2.0f", B[i]);
	}
  }
  printf("\n");
  printf("C (before)\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.0f\n", C[i]);
	}else{
	  printf(" %2.0f", C[i]);
	}
  }
  printf("\n");

  hipMemcpy(dA, A, sizeof(double)*N, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(double)*N, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, sizeof(double)*N, hipMemcpyHostToDevice);

  dim3 grids;
  dim3 blocks;
  grids = dim3(1, 1, 1);
  blocks = dim3(32, 1 ,1);
  hipDeviceSynchronize();
  gpukernel<<<grids,blocks>>>(N, dC, dA, dB);
  hipDeviceSynchronize();
  hipMemcpy(C, dC, sizeof(double)*N, hipMemcpyDeviceToHost);

  printf("C (after)\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.0f\n", C[i]);
	}else{
	  printf(" %2.0f", C[i]);
	}
  }
  printf("\n");

  hipFree(dA); hipFree(dB); hipFree(dC);
  free(A); free(B); free(C);
  return 0;
}
