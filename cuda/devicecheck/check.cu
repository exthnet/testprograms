// -*- c++ -*-

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>

int main(int argc, char **argv)
{
  hipError_t err;
  hipDeviceProp_t dev;
  int nDevices;
  int i;

  err = hipGetDeviceCount(&nDevices);
  /*
  if(err!=cudaSuccess){
	printf("cudaGetDeviceCount failed\n");
	return err;
  }
  */
  printf("%d GPU(s) found\n", nDevices);
  for(i=0; i<nDevices; i++){
	err = hipGetDeviceProperties(&dev, i);
	printf("%d: %s %d\n", i, dev.name, dev.uuid);
  }
  return 0;
}

