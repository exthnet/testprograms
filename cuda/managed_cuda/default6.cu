/* -*- c++ -*- */
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHK_DO(o) if(hipSuccess!=o){printf("%d failed\n",__LINE__);}
#define ALLOCATE(x,y) x=(double*)malloc(sizeof(double)*N*N);CHK_DO(hipMalloc((void**)&y, sizeof(double)*N*N));
//, cudaMemAttachGlobal))

__global__ void gpukernel(int N, double *a, double *b, double*c)
{
  int i, j, k;
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  for(k=0; k<2; k++){
		c[i*N+j] += a[i*N+j] * b[i*N+j];
	  }
	}
  }
}

int main(int argc, char **argv)
{
  hipStream_t s1, s2, s3, s4;
  double *A1, *B1, *C1;
  double *A2, *B2, *C2;
  double *A3, *B3, *C3;
  double *A4, *B4, *C4;
  double *dA1, *dB1, *dC1;
  double *dA2, *dB2, *dC2;
  double *dA3, *dB3, *dC3;
  double *dA4, *dB4, *dC4;
  hipError_t cudaRet;
  int N, M, i, j, k;
  double sum;
  if(argc<2){
	N = 10;
  }else{
	N = atoi(argv[1]);
  }
  printf("N = %d\n", N);
  if(argc<3){
	M = 100;
  }else{
	M = atoi(argv[2]);
  }
  printf("M = %d\n", M);

  ALLOCATE(A1,dA1);
  ALLOCATE(B1,dB1);
  ALLOCATE(C1,dC1);
  ALLOCATE(A2,dA2);
  ALLOCATE(B2,dB2);
  ALLOCATE(C2,dC2);
  ALLOCATE(A3,dA3);
  ALLOCATE(B3,dB3);
  ALLOCATE(C3,dC3);
  ALLOCATE(A4,dA4);
  ALLOCATE(B4,dB4);
  ALLOCATE(C4,dC4);

  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A1[i*N+j] = 1.0;
	  B1[i*N+j] = 2.0;
	  C1[i*N+j] = 0.0;
	}
  }
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A2[i*N+j] = 1.0;
	  B2[i*N+j] = 2.0;
	  C2[i*N+j] = 0.0;
	}
  }
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A3[i*N+j] = 1.0;
	  B3[i*N+j] = 2.0;
	  C3[i*N+j] = 0.0;
	}
  }
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A4[i*N+j] = 1.0;
	  B4[i*N+j] = 2.0;
	  C4[i*N+j] = 0.0;
	}
  }
  CHK_DO(hipStreamCreate(&s1));
  CHK_DO(hipStreamCreate(&s2));
  CHK_DO(hipStreamCreate(&s3));
  CHK_DO(hipStreamCreate(&s4));
  CHK_DO(hipStreamSynchronize(s1));
  CHK_DO(hipStreamSynchronize(s2));
  CHK_DO(hipStreamSynchronize(s3));
  CHK_DO(hipStreamSynchronize(s4));

  #pragma omp parallel sections
  {
  #pragma omp section
  {
  hipMemcpyAsync(dA1, A1, sizeof(double)*N*N, hipMemcpyHostToDevice, s1);
  hipMemcpyAsync(dB1, B1, sizeof(double)*N*N, hipMemcpyHostToDevice, s1);
  hipMemcpyAsync(dC1, C1, sizeof(double)*N*N, hipMemcpyHostToDevice, s1);
  gpukernel<<<M,512,0,s1>>>(N,dA1,dB1,dC1);
  hipMemcpyAsync(A1, dA1, sizeof(double)*N*N, hipMemcpyDeviceToHost, s1);
  hipMemcpyAsync(B1, dB1, sizeof(double)*N*N, hipMemcpyDeviceToHost, s1);
  hipMemcpyAsync(C1, dC1, sizeof(double)*N*N, hipMemcpyDeviceToHost, s1);
  hipMemcpyAsync(dA1, A1, sizeof(double)*N*N, hipMemcpyHostToDevice, s1);
  hipMemcpyAsync(dB1, B1, sizeof(double)*N*N, hipMemcpyHostToDevice, s1);
  hipMemcpyAsync(dC1, C1, sizeof(double)*N*N, hipMemcpyHostToDevice, s1);
  gpukernel<<<M,512,0,s1>>>(N,dA1,dB1,dC1);
  hipMemcpyAsync(A1, dA1, sizeof(double)*N*N, hipMemcpyDeviceToHost, s1);
  hipMemcpyAsync(B1, dB1, sizeof(double)*N*N, hipMemcpyDeviceToHost, s1);
  hipMemcpyAsync(C1, dC1, sizeof(double)*N*N, hipMemcpyDeviceToHost, s1);
  }
#pragma omp section
  {
  hipMemcpyAsync(dA2, A2, sizeof(double)*N*N, hipMemcpyHostToDevice, s2);
  hipMemcpyAsync(dB2, B2, sizeof(double)*N*N, hipMemcpyHostToDevice, s2);
  hipMemcpyAsync(dC2, C2, sizeof(double)*N*N, hipMemcpyHostToDevice, s2);
  gpukernel<<<M,512,0,s2>>>(N,dA2,dB2,dC2);
  hipMemcpyAsync(A2, dA2, sizeof(double)*N*N, hipMemcpyDeviceToHost, s2);
  hipMemcpyAsync(B2, dB2, sizeof(double)*N*N, hipMemcpyDeviceToHost, s2);
  hipMemcpyAsync(C2, dC2, sizeof(double)*N*N, hipMemcpyDeviceToHost, s2);
  }
#pragma omp section
  {
  hipMemcpyAsync(dA4, A4, sizeof(double)*N*N, hipMemcpyHostToDevice, s4);
  hipMemcpyAsync(dB4, B4, sizeof(double)*N*N, hipMemcpyHostToDevice, s4);
  hipMemcpyAsync(dC4, C4, sizeof(double)*N*N, hipMemcpyHostToDevice, s4);
  gpukernel<<<M,512,0,s4>>>(N,dA4,dB4,dC4);
  hipMemcpyAsync(A4, dA4, sizeof(double)*N*N, hipMemcpyDeviceToHost, s4);
  hipMemcpyAsync(B4, dB4, sizeof(double)*N*N, hipMemcpyDeviceToHost, s4);
  hipMemcpyAsync(C4, dC4, sizeof(double)*N*N, hipMemcpyDeviceToHost, s4);
  }
#pragma omp section
  {
  hipMemcpyAsync(dA3, A3, sizeof(double)*N*N, hipMemcpyHostToDevice, s3);
  hipMemcpyAsync(dB3, B3, sizeof(double)*N*N, hipMemcpyHostToDevice, s3);
  hipMemcpyAsync(dC3, C3, sizeof(double)*N*N, hipMemcpyHostToDevice, s3);
  gpukernel<<<M,512,0,s3>>>(N,dA3,dB3,dC3);
  hipMemcpyAsync(A3, dA3, sizeof(double)*N*N, hipMemcpyDeviceToHost, s3);
  hipMemcpyAsync(B3, dB3, sizeof(double)*N*N, hipMemcpyDeviceToHost, s3);
  hipMemcpyAsync(C3, dC3, sizeof(double)*N*N, hipMemcpyDeviceToHost, s3);
  }
}

  CHK_DO(hipStreamSynchronize(s1));
  CHK_DO(hipStreamSynchronize(s2));
  CHK_DO(hipStreamSynchronize(s3));
  CHK_DO(hipStreamSynchronize(s4));
  CHK_DO(hipStreamDestroy(s1));
  CHK_DO(hipStreamDestroy(s2));
  CHK_DO(hipStreamDestroy(s3));
  CHK_DO(hipStreamDestroy(s4));

  sum = 0.0;
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C1[i*N+j];
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C2[i*N+j];
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C3[i*N+j];
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C4[i*N+j];
  printf("sum = %f\n", sum);

  hipFree(dA1);
  hipFree(dB1);
  hipFree(dC1);
  hipFree(dA2);
  hipFree(dB2);
  hipFree(dC2);
  hipFree(dA3);
  hipFree(dB3);
  hipFree(dC3);
  hipFree(dA4);
  hipFree(dB4);
  hipFree(dC4);
  free(A1);
  free(B1);
  free(C1);
  free(A2);
  free(B2);
  free(C2);
  free(A3);
  free(B3);
  free(C3);
  free(A4);
  free(B4);
  free(C4);
  return 0;
}
