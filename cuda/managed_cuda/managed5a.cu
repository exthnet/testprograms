/* -*- c++ -*- */
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHK_DO(o) if(hipSuccess!=o){printf("%d failed\n",__LINE__);}
#define ALLOCATE(x) CHK_DO(hipMallocManaged((void**)&x, sizeof(double)*N*N))
//, cudaMemAttachGlobal))

__global__ void gpukernel(int N, double *a, double *b, double*c)
{
  int i, j, k;
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  for(k=0; k<2; k++){
		c[i*N+j] += a[i*N+j] * b[i*N+j];
	  }
	}
  }
}

int main(int argc, char **argv)
{
  hipStream_t s1, s2, s3, s4;
  double *A1, *B1, *C1;
  double *A2, *B2, *C2;
  double *A3, *B3, *C3;
  double *A4, *B4, *C4;
  hipError_t cudaRet;
  int N, M, i, j, k;
  double sum;
  if(argc<2){
	N = 10;
  }else{
	N = atoi(argv[1]);
  }
  printf("N = %d\n", N);
  if(argc<3){
	M = 100;
  }else{
	M = atoi(argv[2]);
  }
  printf("M = %d\n", M);

  ALLOCATE(A1);
  ALLOCATE(B1);
  ALLOCATE(C1);
  ALLOCATE(A2);
  ALLOCATE(B2);
  ALLOCATE(C2);
  ALLOCATE(A3);
  ALLOCATE(B3);
  ALLOCATE(C3);
  ALLOCATE(A4);
  ALLOCATE(B4);
  ALLOCATE(C4);

  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A1[i*N+j] = 1.0;
	  B1[i*N+j] = 2.0;
	  C1[i*N+j] = 0.0;
	}
  }
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A2[i*N+j] = 1.0;
	  B2[i*N+j] = 2.0;
	  C2[i*N+j] = 0.0;
	}
  }
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A3[i*N+j] = 1.0;
	  B3[i*N+j] = 2.0;
	  C3[i*N+j] = 0.0;
	}
  }
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A4[i*N+j] = 1.0;
	  B4[i*N+j] = 2.0;
	  C4[i*N+j] = 0.0;
	}
  }
  CHK_DO(hipStreamCreate(&s1));
  CHK_DO(hipStreamCreate(&s2));
  CHK_DO(hipStreamCreate(&s3));
  CHK_DO(hipStreamCreate(&s4));
  CHK_DO(hipStreamSynchronize(s1));
  CHK_DO(hipStreamSynchronize(s2));
  CHK_DO(hipStreamSynchronize(s3));
  CHK_DO(hipStreamSynchronize(s4));

  gpukernel<<<M,1024,0,s1>>>(N,A1,B1,C1);
  gpukernel<<<M,1024,0,s2>>>(N,A2,B2,C2);
  gpukernel<<<M,1024,0,s4>>>(N,A4,B4,C4);
  gpukernel<<<M,1024,0,s3>>>(N,A3,B3,C3);
  gpukernel<<<M,1024,0,s1>>>(N,A1,B1,C1);

  CHK_DO(hipStreamSynchronize(s1));
  CHK_DO(hipStreamSynchronize(s2));
  CHK_DO(hipStreamSynchronize(s3));
  CHK_DO(hipStreamSynchronize(s4));
  CHK_DO(hipStreamDestroy(s1));
  CHK_DO(hipStreamDestroy(s2));
  CHK_DO(hipStreamDestroy(s3));
  CHK_DO(hipStreamDestroy(s4));

  sum = 0.0;
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C1[i*N+j];
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C2[i*N+j];
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C3[i*N+j];
  for(i=0; i<N; i++)for(j=0; j<N; j++)sum += C4[i*N+j];
  printf("sum = %f\n", sum);

  hipFree(A1);
  hipFree(B1);
  hipFree(C1);
  hipFree(A2);
  hipFree(B2);
  hipFree(C2);
  hipFree(A3);
  hipFree(B3);
  hipFree(C3);
  hipFree(A4);
  hipFree(B4);
  hipFree(C4);
  return 0;
}
