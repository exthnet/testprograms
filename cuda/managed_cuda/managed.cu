/* -*- c++ -*- */
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHK_DO(o) if(hipSuccess!=o){printf("%d failed\n",__LINE__);}

__global__ void gpukernel(int N, double *a, double *b, double*c)
{
  int i, j, k;
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  for(k=0; k<10; k++){
		c[i*N+j] += a[i*N+j] * b[i*N+j];
	  }
	}
  }
}

int main(int argc, char **argv)
{
  hipStream_t s1;
  double *A1, *B1, *C1;
  hipError_t cudaRet;
  int N, i, j, k;
  if(argc<2){
	N = 10;
  }else{
	N = atoi(argv[1]);
  }
  printf("N = %d\n", N);
  CHK_DO(hipMallocManaged((void**)&A1, sizeof(double)*N*N, hipMemAttachGlobal));
  CHK_DO(hipMallocManaged((void**)&B1, sizeof(double)*N*N, hipMemAttachGlobal));
  CHK_DO(hipMallocManaged((void**)&C1, sizeof(double)*N*N, hipMemAttachGlobal));

  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  A1[i*N+j] = 1.0;
	  B1[i*N+j] = 2.0;
	  C1[i*N+j] = 0.0;
	}
  }
  CHK_DO(hipStreamCreate(&s1));
  CHK_DO(hipStreamSynchronize(s1));
  gpukernel<<<N,N,0,s1>>>(N,A1,B1,C1);

  CHK_DO(hipStreamSynchronize(s1));
  CHK_DO(hipStreamDestroy(s1));

  hipFree(A1);
  hipFree(B1);
  hipFree(C1);
  return 0;
}
