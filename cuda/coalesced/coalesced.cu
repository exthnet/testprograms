// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

/*
64x64行列を32threadsで読み込むテスト
 */

__global__ void gpukernel1(double *mat, double *vec)
{
  int tid = threadIdx.x;
  double tmp;
  int x, y;
  tmp = 0.0;
  for(y=0; y<64; y++){
	for(x=tid; x<64; x+=32){
	  tmp += mat[y*64+x];
	}
  }
  vec[tid] = tmp;
}

__global__ void gpukernel2(double *mat, double *vec)
{
  int tid = threadIdx.x;
  double tmp;
  int x, y;
  tmp = 0.0;
  for(y=tid/16; y<64; y+=2){
	for(x=tid%16; x<64; x+=16){
	  tmp += mat[y*64+x];
	}
  }
  vec[tid] = tmp;
}

__global__ void gpukernel4(double *mat, double *vec)
{
  int tid = threadIdx.x;
  double tmp;
  int x, y;
  tmp = 0.0;
  for(y=tid/8; y<64; y+=4){
	for(x=tid%8; x<64; x+=8){
	  tmp += mat[y*64+x];
	}
  }
  vec[tid] = tmp;
}

__global__ void gpukernel8(double *mat, double *vec)
{
  int tid = threadIdx.x;
  double tmp;
  int x, y;
  tmp = 0.0;
  for(y=tid/4; y<64; y+=8){
	for(x=tid%4; x<64; x+=4){
	  tmp += mat[y*64+x];
	}
  }
  vec[tid] = tmp;
}

__global__ void gpukernel16(double *mat, double *vec)
{
  int tid = threadIdx.x;
  double tmp;
  int x, y;
  tmp = 0.0;
  for(y=tid/2; y<64; y+=16){
	for(x=tid%2; x<64; x+=2){
	  tmp += mat[y*64+x];
	}
  }
  vec[tid] = tmp;
}

__global__ void gpukernel32(double *mat, double *vec)
{
  int tid = threadIdx.x;
  double tmp;
  int x, y;
  tmp = 0.0;
  for(y=tid; y<64; y+=32){
	for(x=0; x<64; x+=1){
	  tmp += mat[y*64+x];
	}
  }
  vec[tid] = tmp;
}

template <int X>
__global__ void gpukernelX(double *mat, double *vec)
{
  int tid = threadIdx.x;
  double tmp;
  int x, y;
  tmp = 0.0;
  for(y=tid/X; y<64; y+=(32/X)){
	for(x=tid%X; x<64; x+=X){
	  tmp += mat[y*64+x];
	}
  }
  vec[tid] = tmp;
}

void gpukernel_driver(int x, double *dmat, double *dvec)
{
  switch(x){
  case 1:  gpukernelX< 1><<<1,32>>>(dmat, dvec); break;
  case 2:  gpukernelX< 2><<<1,32>>>(dmat, dvec); break;
  case 4:  gpukernelX< 4><<<1,32>>>(dmat, dvec); break;
  case 8:  gpukernelX< 8><<<1,32>>>(dmat, dvec); break;
  case 16: gpukernelX<16><<<1,32>>>(dmat, dvec); break;
  case 32: gpukernelX<32><<<1,32>>>(dmat, dvec); break;
  }
}

int main(int argc, char **argv)
{
  int i, N;
  double *mat, *vec;
  double *dmat, *dvec;
  double d;

  N = 64;
  mat = (double*)malloc(sizeof(double)*N*N);
  vec = (double*)malloc(sizeof(double)*N);

  for(i=0;i<N*N;i++){
	mat[i] = (double)(i+1);
  }
  for(i=0;i<N;i++){
	vec[i] = 0.0f;
  }

  hipMalloc((void**)&dmat, sizeof(double)*N*N);
  hipMalloc((void**)&dvec, sizeof(double)*N);

  hipMemcpy(dmat, mat, sizeof(double)*N*N, hipMemcpyHostToDevice);

  // 1
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernel1<<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", 1, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 2
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernel2<<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", 2, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 4
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernel4<<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", 4, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 8
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernel8<<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", 8, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 16
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernel16<<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", 16, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 32
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernel32<<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", 32, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);




  int X;

  // 1
  X=32;
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernelX<32><<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", X, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 2
  X=16;
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernelX<16><<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", X, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 4
  X=8;
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernelX<8><<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", X, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 8
  X=4;
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernelX<4><<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", X, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 16
  X=2;
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernelX<2><<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", X, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // 32
  X=1;
  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernelX<1><<<1,32>>>(dmat, dvec);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;
  printf("%2d: time %f, ", X, d);
  hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
  d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);

  // driver
  for(X=32;X>=1;X>>=1){
	d=omp_get_wtime();
	for(i=0;i<100;i++){
	  hipDeviceSynchronize();
	  gpukernel_driver(X,dmat, dvec);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;
	printf("%2d: time %f, ", X, d);
	hipMemcpy(vec, dvec, sizeof(double)*N, hipMemcpyDeviceToHost);
	d=0; for(i=0;i<32;i++)d+=vec[i]; printf("d=%f\n",d);
  }

  for(i=0;i<N;i++){
	printf(" %f", vec[i]);
  }
  printf("\n");

  hipFree(dmat); hipFree(dvec);
  free(mat); free(vec);
  return 0;
}
