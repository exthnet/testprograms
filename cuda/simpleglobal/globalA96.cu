// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define NEXEC 1
// number of warp
#define NW 3

__global__ void gpukernel_32a(double *out, double *in, int N, int *head)
{
  int n;
  int y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/32; n<N; n+=NW){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y++){
	  tmp += in[y*32 + threadIdx.x%32 + h*32*32];
	}
	if(threadIdx.x%32==0)out[n] += tmp;
  }
}
__global__ void gpukernel_32b(double *out, double *in, int N, int *head)
{
  int n;
  int x;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/32; n<N; n+=NW){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x++){
	  tmp += in[(threadIdx.x%32)*32 + x + h*32*32];
	}
	if(threadIdx.x%32==0)out[n] += tmp;
  }
}

__global__ void gpukernel_16aa(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/16; n<N; n+=NW*2){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y++){
	  for(x=0; x<32; x+=16){
		tmp += in[y*32 + threadIdx.x%16+x + h*32*32];
	  }
	}
	if(threadIdx.x%16==0)out[n] += tmp;
  }
}
__global__ void gpukernel_16ab(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/16; n<N; n+=NW*2){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=16){
	  for(y=0; y<32; y++){
		tmp += in[y*32 + threadIdx.x%16+x + h*32*32];
	  }
	}
	if(threadIdx.x%16==0)out[n] += tmp;
  }
}
__global__ void gpukernel_16ba(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/16; n<N; n+=NW*2){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y+=16){
	  for(x=0; x<32; x+=1){
		tmp += in[(y+threadIdx.x%16)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%16==0)out[n] += tmp;
  }
}
__global__ void gpukernel_16bb(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/16; n<N; n+=NW*2){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=1){
	  for(y=0; y<32; y+=16){
		tmp += in[(y+threadIdx.x%16)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%16==0)out[n] += tmp;
  }
}

__global__ void gpukernel_8aa(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/8; n<N; n+=NW*4){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y++){
	  for(x=0; x<32; x+=8){
		tmp += in[y*32 + threadIdx.x%8+x + h*32*32];
	  }
	}
	if(threadIdx.x%8==0)out[n] += tmp;
  }
}
__global__ void gpukernel_8ab(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/8; n<N; n+=NW*4){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=8){
	  for(y=0; y<32; y++){
		tmp += in[y*32 + threadIdx.x%8+x + h*32*32];
	  }
	}
	if(threadIdx.x%8==0)out[n] += tmp;
  }
}
__global__ void gpukernel_8ba(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/8; n<N; n+=NW*4){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y+=8){
	  for(x=0; x<32; x+=1){
		tmp += in[(y+threadIdx.x%8)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%8==0)out[n] += tmp;
  }
}
__global__ void gpukernel_8bb(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/8; n<N; n+=NW*4){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=1){
	  for(y=0; y<32; y+=8){
		tmp += in[(y+threadIdx.x%8)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%8==0)out[n] += tmp;
  }
}

__global__ void gpukernel_4aa(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/4; n<N; n+=NW*8){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y++){
	  for(x=0; x<32; x+=4){
		tmp += in[y*32 + threadIdx.x%4+x + h*32*32];
	  }
	}
	if(threadIdx.x%4==0)out[n] += tmp;
  }
}
__global__ void gpukernel_4ab(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/4; n<N; n+=NW*8){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=4){
	  for(y=0; y<32; y++){
		tmp += in[y*32 + threadIdx.x%4+x + h*32*32];
	  }
	}
	if(threadIdx.x%4==0)out[n] += tmp;
  }
}
__global__ void gpukernel_4ba(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/4; n<N; n+=NW*8){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y+=4){
	  for(x=0; x<32; x+=1){
		tmp += in[(y+threadIdx.x%4)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%4==0)out[n] += tmp;
  }
}
__global__ void gpukernel_4bb(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/4; n<N; n+=NW*8){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=1){
	  for(y=0; y<32; y+=4){
		tmp += in[(y+threadIdx.x%4)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%4==0)out[n] += tmp;
  }
}

__global__ void gpukernel_2aa(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/2; n<N; n+=NW*16){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y++){
	  for(x=0; x<32; x+=2){
		tmp += in[y*32 + threadIdx.x%2+x + h*32*32];
	  }
	}
	if(threadIdx.x%2==0)out[n] += tmp;
  }
}
__global__ void gpukernel_2ab(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/2; n<N; n+=NW*16){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=2){
	  for(y=0; y<32; y++){
		tmp += in[y*32 + threadIdx.x%2+x + h*32*32];
	  }
	}
	if(threadIdx.x%2==0)out[n] += tmp;
  }
}
__global__ void gpukernel_2ba(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/2; n<N; n+=NW*16){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y+=2){
	  for(x=0; x<32; x+=1){
		tmp += in[(y+threadIdx.x%2)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%2==0)out[n] += tmp;
  }
}
__global__ void gpukernel_2bb(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x/2; n<N; n+=NW*16){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=1){
	  for(y=0; y<32; y+=2){
		tmp += in[(y+threadIdx.x%2)*32 + x + h*32*32];
	  }
	}
	if(threadIdx.x%2==0)out[n] += tmp;
  }
}

__global__ void gpukernel_1a(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x; n<N; n+=NW*32){
	h = head[n];
	tmp = 0.0;
	for(y=0; y<32; y++){
	  for(x=0; x<32; x+=1){
		tmp += in[y*32 + x + h*32*32];
	  }
	}
	out[n] += tmp;
  }
}
__global__ void gpukernel_1b(double *out, double *in, int N, int *head)
{
  int n;
  int x, y;
  double tmp = 0.0;
  int h;
  for(n=threadIdx.x; n<N; n+=NW*32){
	h = head[n];
	tmp = 0.0;
	for(x=0; x<32; x+=1){
	  for(y=0; y<32; y++){
		tmp += in[y*32 + x + h*32*32];
	  }
	}
	out[n] += tmp;
  }
}

// ######## ######## ######## ######## ######## ######## ######## ########

void swap(int *a, int *b)
{
  int x = *a;
  *a = *b;
  *b = x;
}

int main(int argc, char **argv)
{
  int rule = 0;
  int N = 320000;
  int len = 32 * 32 * N;
  int i, x;
  double *out, *in;
  double *dout, *din;
  double d;
  int *head, *dhead;

  if(argc>1)rule=atoi(argv[1]); printf("rule=%d\n", rule);
  //if(argc>1)N=atoi(argv[1]); printf("N=%d\n", N);
  out = (double*)malloc(sizeof(double)*32*N);
  in = (double*)malloc(sizeof(double)*len);
  head = (int*)malloc(sizeof(int)*N);

  hipMalloc((void**)&dout, sizeof(double)*32*N);
  hipMalloc((void**)&din, sizeof(double)*len);
  hipMalloc((void**)&dhead, sizeof(int)*N);

#define BENCH(FUNCNAME,X) \
  {\
	for(i=0;i<32*N;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<len;i++){\
	  in[i] = (double)(i+1)/1000.0;\
	}\
	if(rule==0)for(i=0;i<N;i++)head[i] = i;\
	if(rule==1)for(i=0;i<N;i++)head[i] = N-1-i;\
	if(rule==2){\
	  for(i=0;i<N;i++)head[i] = i;\
	  for(i=0;i<N;i++)swap(&head[rand()%N], &head[rand()%N]);\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32*N, hipMemcpyHostToDevice);\
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dhead, head, sizeof(int)*N, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  FUNCNAME<<<1,32*NW>>>(dout, din, N, dhead);\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32*N, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  FUNCNAME<<<1,32*NW>>>(dout, din, N, dhead);\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", X, d);\
	d=0.0; for(i=0;i<32*N;i++)d+=out[i]; printf("d=%.2f\n",d);\
  }

  BENCH(gpukernel_1a ,  "  1a ");
  BENCH(gpukernel_1b ,  "  1b ");
  BENCH(gpukernel_32a,  " 32a ");
  BENCH(gpukernel_32b,  " 32b ");
  BENCH(gpukernel_16aa, " 16aa");
  BENCH(gpukernel_16ab, " 16ab");
  BENCH(gpukernel_16ba, " 16ba");
  BENCH(gpukernel_16bb, " 16bb");
  BENCH(gpukernel_8aa,  "  8aa");
  BENCH(gpukernel_8ab,  "  8ab");
  BENCH(gpukernel_8ba,  "  8ba");
  BENCH(gpukernel_8bb,  "  8bb");
  BENCH(gpukernel_4aa,  "  4aa");
  BENCH(gpukernel_4ab,  "  4ab");
  BENCH(gpukernel_4ba,  "  4ba");
  BENCH(gpukernel_4bb,  "  4bb");
  BENCH(gpukernel_2aa,  "  2aa");
  BENCH(gpukernel_2ab,  "  2ab");
  BENCH(gpukernel_2ba,  "  2ba");
  BENCH(gpukernel_2bb,  "  2bb");
  BENCH(gpukernel_1a,   "  1a ");
  BENCH(gpukernel_1b,   "  1b ");

  hipFree(dout);  hipFree(din);
  free(out); free(in);
  return 0;
}
