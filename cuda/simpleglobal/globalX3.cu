// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define NLOOP 1000
#define NEXEC 1000
#define IEXEC 1000

/*
W=32
  0  1  2  3 ... 31
W=16
  0  1  2  3 ... 15
 16 17 18 19 ... 31
W=8
  0  1  2  3 ...  7
  8  9 10 11 ... 15
 16 17 18 19 ... 23
 24 25 26 27 ... 31
W=4
  0  1  2  3
  4  5  6  7
  ...
 28 29 30 31
W=2
  0  1
  2  3
 ...
 30 31
W=1
  0
  1
  2
  3
 ...
 31
*/

__global__ void gpukernelW32(double *out, double *in, int iloop)
{
  int i, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y++){
	  tmp += in[y*32+threadIdx.x + i*32*32];
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW16a(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=2){
	  for(x=0; x<32; x+=16){
		tmp += in[(y+threadIdx.x/16)*32 + threadIdx.x%16+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW16b(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=16){
	  for(y=0; y<32; y+=2){
		tmp += in[(y+threadIdx.x/16)*32 + threadIdx.x%16+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW8a(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=4){
	  for(x=0; x<32; x+=8){
		tmp += in[(y+threadIdx.x/8)*32 + threadIdx.x%8+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW8b(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=8){
	  for(y=0; y<32; y+=4){
		tmp += in[(y+threadIdx.x/8)*32 + threadIdx.x%8+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW4a(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=8){
	  for(x=0; x<32; x+=4){
		tmp += in[(y+threadIdx.x/4)*32 + threadIdx.x%4+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW4b(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=4){
	  for(y=0; y<32; y+=8){
		tmp += in[(y+threadIdx.x/4)*32 + threadIdx.x%4+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW2a(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=16){
	  for(x=0; x<32; x+=2){
		tmp += in[(y+threadIdx.x/2)*32 + threadIdx.x%2+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW2b(double *out, double *in, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=2){
	  for(y=0; y<32; y+=16){
		tmp += in[(y+threadIdx.x/2)*32 + threadIdx.x%2+x + i*32*32];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernelW1(double *out, double *in, int iloop)
{
  int i, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x++){
	  tmp += in[threadIdx.x*32+x + i*32*32];
	}
  }
  out[threadIdx.x] += tmp;
}

// general
template<int N>
__global__ void gpukernelWNa(double *out, double *in, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=32/N){
	  for(x=0; x<32; x+=N){
		tmp += in[(y+threadIdx.x/N)*32 + threadIdx.x%N+x + i*32*32];// * vec[x];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}
template<int N>
__global__ void gpukernelWNb(double *out, double *in, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=N){
	  for(y=0; y<32; y+=32/N){
		tmp += in[(y+threadIdx.x/N)*32 + threadIdx.x%N+x + i*32*32];// * vec[x];
	  }
	}
  }
  out[threadIdx.x] += tmp;
}

// ######## ######## ######## ######## ######## ######## ######## ########

int main(int argc, char **argv)
{
  int len = NLOOP * 32 * 32;
  int i, x;
  double *out, *in;
  double *dout, *din;
  double d;

  out = (double*)malloc(sizeof(double)*32);
  in = (double*)malloc(sizeof(double)*len);

  hipMalloc((void**)&dout, sizeof(double)*32);
  hipMalloc((void**)&din, sizeof(double)*len);

#define BENCH1(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<len;i++){\
	  in[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<<<1,32>>>(dout, din, ILOOP);				\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<<<1,32>>>(dout, din, ILOOP);				\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);\
  }

#define BENCH2(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<len;i++){\
	  in[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWN<KERNEL><<<1,32>>>(dout, din, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWN<KERNEL><<<1,32>>>(dout, din, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);\
  }
#define BENCH2a(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<len;i++){\
	  in[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNa<KERNEL><<<1,32>>>(dout, din, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNa<KERNEL><<<1,32>>>(dout, din, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);\
  }
#define BENCH2b(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<len;i++){\
	  in[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNb<KERNEL><<<1,32>>>(dout, din, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNb<KERNEL><<<1,32>>>(dout, din, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);\
  }

  for(x=0;x<2;x++){
	BENCH1(gpukernelW32,  " W32", 1);
	BENCH1(gpukernelW16a, "W16r", 1);
	BENCH1(gpukernelW16b, "W16c", 1);
	BENCH1(gpukernelW8a,  " W8r", 1);
	BENCH1(gpukernelW8b,  " W8c", 1);
	BENCH1(gpukernelW4a,  " W4r", 1);
	BENCH1(gpukernelW4b,  " W4c", 1);
	BENCH1(gpukernelW2a,  " W2r", 1);
	BENCH1(gpukernelW2b,  " W2c", 1);
	BENCH1(gpukernelW1,   "  W1", 1);
  }
  for(x=0;x<2;x++){
	BENCH1(gpukernelW32,  " W32", IEXEC);
	BENCH1(gpukernelW16a, "W16r", IEXEC);
	BENCH1(gpukernelW16b, "W16c", IEXEC);
	BENCH1(gpukernelW8a,  " W8r", IEXEC);
	BENCH1(gpukernelW8b,  " W8c", IEXEC);
	BENCH1(gpukernelW4a,  " W4r", IEXEC);
	BENCH1(gpukernelW4b,  " W4c", IEXEC);
	BENCH1(gpukernelW2a,  " W2r", IEXEC);
	BENCH1(gpukernelW2b,  " W2c", IEXEC);
	BENCH1(gpukernelW1,   "  W1", IEXEC);
  }

  for(x=0;x<2;x++){
	BENCH2a(32,  "W32r", IEXEC);
	BENCH2b(32,  "W32c", IEXEC);
	BENCH2a(32,  "W32r", IEXEC);
	BENCH2b(32,  "W32c", IEXEC);
	BENCH2a(16,  "W16r", IEXEC);
	BENCH2b(16,  "W16c", IEXEC);
	BENCH2a(8,   " W8r", IEXEC);
	BENCH2b(8,   " W8c", IEXEC);
	BENCH2a(4,   " W4r", IEXEC);
	BENCH2b(4,   " W4c", IEXEC);
	BENCH2a(2,   " W2r", IEXEC);
	BENCH2b(2,   " W2c", IEXEC);
	BENCH2a(1,   " W1r", IEXEC);
	BENCH2b(1,   " W1c", IEXEC);
  }

  hipFree(dout);  hipFree(din);
  free(out); free(in);
  return 0;
}
