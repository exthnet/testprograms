// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define NEXEC 1000

__global__ void gpukernelW32(double *out, double *mat, double *vec)
{
  int y;
  double tmp = 0.0;
  for(y=0; y<32; y++){
	tmp = mat[y*32+threadIdx.x] * vec[threadIdx.x];
	for(int offset=16; offset>0; offset/=2){
	  tmp += __shfl_down_sync
		(0xffffffff, tmp, offset, 32);
	}
	if(threadIdx.x==0)out[threadIdx.x] += tmp;
  }
}

__global__ void gpukernelW1(double *out, double *mat, double *vec)
{
  int x;
  double tmp = 0.0;
  for(x=0; x<32; x++){
	tmp += mat[threadIdx.x*32+x] * vec[x];
  }
  out[threadIdx.x] += tmp;
}

// ######## ######## ######## ######## ######## ######## ######## ########

int main(int argc, char **argv)
{
  int N = 1000;
  int len = 32 * 32;
  int i, x;
  double *out, *mat, *vec;
  double *dout, *dmat, *dvec;
  double d;

  if(argc>1)N=atoi(argv[1]); printf("N=%d\n", N);
  out = (double*)malloc(sizeof(double)*32);
  mat = (double*)malloc(sizeof(double)*len);
  vec = (double*)malloc(sizeof(double)*32);

  hipMalloc((void**)&dout, sizeof(double)*32);
  hipMalloc((void**)&dmat, sizeof(double)*len);
  hipMalloc((void**)&dvec, sizeof(double)*32);

  {
	x = 32;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	  vec[i] = sin((double)i/10.0);
	}
	for(i=0;i<len;i++){
	  mat[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW32<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW32<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);

  }

  {
	x = 1;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	  vec[i] = sin((double)i/10.0);
	}
	for(i=0;i<len;i++){
	  mat[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW1<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW1<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }

  {
	x = 32;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	  vec[i] = sin((double)i/10.0);
	}
	for(i=0;i<len;i++){
	  mat[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW32<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW32<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }

  {
	x = 1;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	  vec[i] = sin((double)i/10.0);
	}
	for(i=0;i<len;i++){
	  mat[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW1<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernelW1<<<1,32>>>(dout, dmat, dvec);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }


  hipFree(dout); hipFree(dmat); hipFree(dvec);
  free(out); free(mat); free(vec);
  return 0;
}
