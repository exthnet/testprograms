// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

__global__ void gpukernel1(double *out, double *in, int N, int A, int B)
{
  int id;
  int i;
  int begin = (threadIdx.x/B)*(N/A)+(threadIdx.x%B);
  int end = ((threadIdx.x/B)+1)*(N/A);
  int step = B;
  double tmp = 0.0;
  for(id=0;id<1000;id++){
	for(i=begin;i<end;i+=step){
	  tmp += in[id*32+i];
	}
  }
  out[threadIdx.x] += tmp;
}

// ######## ######## ######## ######## ######## ######## ######## ########

int main(int argc, char **argv)
{
  int N = 1000;
  int len = N * 32;
  int i;
  double *out, *in;
  double *dout, *din;
  double d;

  if(argc>1)N=atoi(argv[1]); printf("N=%d\n", N);
  out = (double*)malloc(sizeof(double)*32);
  in = (double*)malloc(sizeof(double)*len);

  hipMalloc((void**)&dout, sizeof(double)*32);
  hipMalloc((void**)&din, sizeof(double)*len);

  int A, B, x;
  for(x=1;x<=32;x*=2){
	A = x; B = 32/x;
	for(i=0;i<32;i++){
	  out[i] = 0.0;
	}
	for(i=0;i<len;i++){
	  in[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);

	for(i=0;i<10;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, 32, A, B);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<10;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, 32, A, B);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }

  for(x=1;x<=32;x*=2){
	A = x; B = 32/x;
	for(i=0;i<32;i++){
	  out[i] = 0.0;
	}
	for(i=0;i<len;i++){
	  in[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);

	for(i=0;i<10;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, 32, A, B);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<10;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, 32, A, B);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }


  hipFree(dout);  hipFree(din);
  free(out); free(in);
  return 0;
}
