// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define NLOOP 1000
#define NEXEC 10
#define IEXEC NLOOP

//#define KERNEL32

// ######## ######## ######## ######## ######## ######## ######## ########
// general WNr, WNc, WNr2, WNc2

template<int N>
__global__ void gpukernelWNr(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=32/N){
	  tmp = 0.0;
	  for(x=0; x<32; x+=N){
		tmp += mat[(y+threadIdx.x/N)*32 + threadIdx.x%N+x + i*32*32] * vec[threadIdx.x%N+x];
	  }
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y+threadIdx.x/N + i*32] += tmp;
	}
  }
}

template<int N, int M>
__global__ void gpukernelAMWNr(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i+=M){
	for(y=0; y<32; y+=32/N){
	  tmp = 0.0;
	  for(x=0; x<32; x+=N){
		tmp += mat[(y+(threadIdx.x%32)/N)*32 + threadIdx.x%N+x + (i+threadIdx.x/32)*32*32];// * vec[threadIdx.x%N+x];
	  }
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y+(threadIdx.x%32)/N + i*32] += tmp;
	}
  }
}
template<int N, int M>
__global__ void gpukernelAMWNr2(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/32;i<iloop;i+=M){
	for(y=(threadIdx.x%32)/N; y<32; y+=32/N){
	  tmp = 0.0;
	  for(x=threadIdx.x%N; x<32; x+=N){
		tmp += mat[y*32 + x + i*32*32];// * vec[threadIdx.x%N+x];
	  }
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y + i*32] += tmp;
	}
  }
}
template<int N, int M>
__global__ void gpukernelAMWNr3(double *out, double *mat, double *vec, int iloop, int w, int h)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/32;i<iloop;i+=M){
	for(y=(threadIdx.x%32)/N; y<h; y+=32/N){
	  tmp = 0.0;
	  for(x=threadIdx.x%N; x<w; x+=N){
		tmp += mat[y*w + x + i*w*h];// * vec[threadIdx.x%N+x];
	  }
	  /*
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y + i*h] += tmp;
	  */
	  atomicAdd(&out[y + i*h], tmp);
	}
  }
}
template<int N, int M>
__global__ void gpukernelAMWNc2(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/32;i<iloop;i+=M){
	for(x=threadIdx.x%N; x<32; x+=N){
	  for(y=(threadIdx.x%32)/N; y<32; y+=32/N){
		tmp = 0.0;
		tmp += mat[y*32 + x + i*32*32];// * vec[threadIdx.x%N+x];
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + i*32] += tmp;
	  }
	}
  }
}
template<int N, int M>
__global__ void gpukernelAMWNc3(double *out, double *mat, double *vec, int iloop, int w, int h)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/32;i<iloop;i+=M){
	for(x=threadIdx.x%N; x<w; x+=N){
	  for(y=(threadIdx.x%32)/N; y<h; y+=32/N){
		tmp = 0.0;
		tmp += mat[y*w + x + i*w*h];// * vec[threadIdx.x%N+x];
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + i*h] += tmp;
	  }
	}
  }
}

template<int N, int M>
__global__ void gpukernelBMWNr(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i+=M*32/N){
	for(y=0; y<32; y+=1){
	  tmp = 0.0;
	  for(x=0; x<32; x+=N){
		tmp += mat[y*32 + threadIdx.x%N+x + (i+threadIdx.x/N)*32*32];// * vec[threadIdx.x%N+x];
	  }
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y + i*32] += tmp;
	}
  }
}
template<int N, int M>
__global__ void gpukernelBMWNr2(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/N;i<iloop;i+=M*32/N){
	for(y=0; y<32; y+=1){
	  tmp = 0.0;
	  for(x=threadIdx.x%N; x<32; x+=N){
		tmp += mat[y*32 + x + i*32*32];// * vec[threadIdx.x%N+x];
	  }
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y + i*32] += tmp;
	}
  }
}
template<int N, int M>
__global__ void gpukernelBMWNr3(double *out, double *mat, double *vec, int iloop, int w, int h)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/N;i<iloop;i+=M*32/N){
	for(y=0; y<h; y+=1){
	  tmp = 0.0;
	  for(x=threadIdx.x%N; x<w; x+=N){
		tmp += mat[y*w + x + i*w*h];// * vec[threadIdx.x%N+x];
	  }
	  /*
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y + i*h] += tmp;
	  */
	  atomicAdd(&out[y + i*h], tmp);
	}
  }
}
template<int N, int M>
__global__ void gpukernelBMWNc2(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/N;i<iloop;i+=M*32/N){
	for(x=threadIdx.x%N; x<32; x+=N){
	  for(y=0; y<32; y+=1){
		tmp = 0.0;
		tmp += mat[y*32 + x + i*32*32];// * vec[threadIdx.x%N+x];
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + i*32] += tmp;
	  }
	}
  }
}
template<int N, int M>
__global__ void gpukernelBMWNc3(double *out, double *mat, double *vec, int iloop, int w, int h)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=threadIdx.x/N;i<iloop;i+=M*32/N){
	for(x=threadIdx.x%N; x<w; x+=N){
	  for(y=0; y<h; y+=1){
		tmp = 0.0;
		tmp += mat[y*w + x + i*w*h];// * vec[threadIdx.x%N+x];
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + i*h] += tmp;
	  }
	}
  }
}

template<int N, int M, int L>
__global__ void gpukernelCLMWNr2(double *out, double *mat, double *vec, int iloop)
{
	// complete
	int i, x, y;
	double tmp = 0.0;
	for(i=threadIdx.x/(N*L);i<iloop;i+=(32*M)/(N*L)){
	  for(y=(threadIdx.x%(N*L))/N; y<32; y+=L){
		tmp = 0.0;
		for(x=threadIdx.x%N; x<32; x+=N){
		  tmp += mat[y*32 + x + i*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + i*32] += tmp;
	  }
	}
}
template<int N, int M, int L>
__global__ void gpukernelCLMWNr3(double *out, double *mat, double *vec, int iloop, int w, int h)
{
	// complete
	int i, x, y;
	double tmp = 0.0;
	for(i=threadIdx.x/(N*L);i<iloop;i+=(32*M)/(N*L)){
	  for(y=(threadIdx.x%(N*L))/N; y<h; y+=L){
		tmp = 0.0;
		for(x=threadIdx.x%N; x<w; x+=N){
		  tmp += mat[y*w + x + i*w*h];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + i*h] += tmp;
	  }
	}
}

template<int N, int M, int L>
__global__ void gpukernelCLMWNr(double *out, double *mat, double *vec, int iloop)
{
#if 0
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i+=M/L){
	for(y=0; y<32; y+=32/N*L){
	  tmp = 0.0;
	  for(x=0; x<32; x+=N){
		tmp += mat[(y+threadIdx.x/(32*(M/L)))*32 + threadIdx.x%N+x + (i+threadIdx.x/(N*L))*32*32];// * vec[threadIdx.x%N+x];
	  }
	  for(int offset=N/2; offset>0; offset/=2){
		tmp += __shfl_down_sync
		  (0xffffffff, tmp, offset, 32);
	  }
	  if(threadIdx.x%N==0)out[y+threadIdx.x/(32*(M/L)) + (i+threadIdx.x/(N*L))*32] += tmp;
	}
  }
#endif
#if 0
  if(N==32){
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=M/L){
	  for(y=0; y<32; y+=L){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y+threadIdx.x%(32*L)/32)*32 + threadIdx.x%N+x + (i+threadIdx.x/(32*L))*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y+threadIdx.x%(32*L)/32 + (i+threadIdx.x/(32*L))*32] += tmp;
	  }
	}
  }else if(N==16){
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=(32*M)/(N*L)){
	  for(y=0; y<32; y+=L){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y+threadIdx.x%(N*L)/N)*32 + threadIdx.x%N+x + (i+threadIdx.x/(N*L))*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y+threadIdx.x%(N*L)/N + (i+threadIdx.x/(N*L))*32] += tmp;
	  }
	}
  }
#endif

#if 0 // correct, notice for overrun
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=(32*M)/(N*L)){
	  for(y=0; y<32; y+=L){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y+(threadIdx.x%(N*L))/N)*32 + threadIdx.x%N+x + (i+threadIdx.x/(N*L))*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y+(threadIdx.x%(N*L))/N + (i+threadIdx.x/(N*L))*32] += tmp;
	  }
	}
#endif

#if 1
	// complete
	int i, x, y;
	double tmp = 0.0;
	for(i=threadIdx.x/(N*L);i<iloop;i+=(32*M)/(N*L)){
	  for(y=(threadIdx.x%(N*L))/N; y<32; y+=L){
		tmp = 0.0;
		for(x=threadIdx.x%N; x<32; x+=N){
		  tmp += mat[y*32 + x + i*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + i*32] += tmp;
	  }
	}
#endif

#if 0
	// open loop: no performance differences
	int i, x, y;
	double tmp = 0.0;
	for(i=threadIdx.x/(N*L);i<iloop;i+=(32*M)/(N*L)){
	  for(y=(threadIdx.x%(N*L))/N; y<32; y+=L){
		tmp = 0.0;
		for(x=threadIdx.x%N; x<32; x+=N){
		  tmp += mat[y*32 + x + i*32*32];// * vec[threadIdx.x%N+x];
		}
		if(N>=32)tmp += __shfl_down_sync(0xffffffff, tmp, 16, 32);
		if(N>=16)tmp += __shfl_down_sync(0xffffffff, tmp,  8, 32);
		if(N>= 8)tmp += __shfl_down_sync(0xffffffff, tmp,  4, 32);
		if(N>= 4)tmp += __shfl_down_sync(0xffffffff, tmp,  2, 32);
		if(N>= 2)tmp += __shfl_down_sync(0xffffffff, tmp,  1, 32);
		if(threadIdx.x%N==0)out[y + i*32] += tmp;
	  }
	}
#endif

#if 0
  if(M==2&&L==1){
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=2){
	  for(y=0; y<32; y+=1){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y)*32 + threadIdx.x%N+x + (i+threadIdx.x/32)*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y + (i+threadIdx.x/32)*32] += tmp;
	  }
	}
  }
  if(M==2&&L==2){
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=1){
	  for(y=0; y<32; y+=2){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y+threadIdx.x/32)*32 + threadIdx.x%N+x + (i)*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y+(threadIdx.x/32) + i*32] += tmp;
	  }
	}
  }
  // M=4, 128threads/block
  if(M==4&&L==1){
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=M/L){
	  for(y=0; y<32; y+=L){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y+threadIdx.x%(32*L)/32)*32 + threadIdx.x%N+x + (i+threadIdx.x/(32*L))*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y+threadIdx.x%(32*L)/32 + (i+threadIdx.x/(32*1))*32] += tmp;
	  }
	}
  }
  if(M==4&&L==2){
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=M/L){
	  for(y=0; y<32; y+=L){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y+threadIdx.x%(32*L)/32)*32 + threadIdx.x%N+x + (i+threadIdx.x/(32*L))*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y+threadIdx.x%(32*L)/32 + (i+threadIdx.x/(32*L))*32] += tmp;
	  }
	}
  }
  if(M==4&&L==4){
	int i, x, y;
	double tmp = 0.0;
	for(i=0;i<iloop;i+=M/L){
	  for(y=0; y<32; y+=L){
		tmp = 0.0;
		for(x=0; x<32; x+=N){
		  tmp += mat[(y+threadIdx.x%(32*L)/32)*32 + threadIdx.x%N+x + (i+threadIdx.x/(32*L))*32*32];// * vec[threadIdx.x%N+x];
		}
		for(int offset=N/2; offset>0; offset/=2){
		  tmp += __shfl_down_sync
			(0xffffffff, tmp, offset, 32);
		}
		if(threadIdx.x%N==0)out[y+threadIdx.x%(32*L)/32 + (i+threadIdx.x/(32*L))*32] += tmp;
	  }
	}
  }
#endif
}

// ######## ######## ######## ######## ######## ######## ######## ########

void swap(int *a, int *b)
{
  int x = *a;
  *a = *b;
  *b = x;
}

int main(int argc, char **argv)
{
  int len = 32 * 32 * NLOOP;
  int i, x;
  double *out, *mat, *vec;
  double *dout, *dmat, *dvec;
  double d;
  int w, h;

  w = h = 32;
  if(argc==3){
	w = atoi(argv[1]);
	h = atoi(argv[2]);
	printf("w=%d, h=%d\n",w,h); fflush(stdout);
  }
  len = w * h * NLOOP;

  out = (double*)malloc(sizeof(double)*h*NLOOP);
  mat = (double*)malloc(sizeof(double)*len);
  vec = (double*)malloc(sizeof(double)*w);

  hipMalloc((void**)&dout, sizeof(double)*h*NLOOP);
  hipMalloc((void**)&dmat, sizeof(double)*len);
  hipMalloc((void**)&dvec, sizeof(double)*w);

#define BENCH1(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<h*NLOOP;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<w;i++){\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/100.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*h*NLOOP, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*w, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<<<1,32>>>(dout, dmat, dvec, ILOOP);				\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*h*NLOOP, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<<<1,32>>>(dout, dmat, dvec, ILOOP);				\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<h*NLOOP;i++)d+=out[i]*(double)(i+1)/10000.0; printf("d=%.2f\n",d); fflush(stdout);\
  }

#define BENCH2(KERNEL,N,NAME,ILOOP)				\
  {\
	for(i=0;i<h*NLOOP;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<w;i++){\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/100.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*h*NLOOP, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*w, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*h*NLOOP, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<h*NLOOP;i++)d+=out[i]*(double)(i+1)/10000.0; printf("d=%.2f\n",d); fflush(stdout); \
  }

#define BENCH30(KERNEL,N,M,NAME,ILOOP)			\
  {\
	for(i=0;i<h*NLOOP;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<w;i++){\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/100.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*h*NLOOP, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*w, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M><<<1,32*M>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*h*NLOOP, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M><<<1,32*M>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<h*NLOOP;i++)d+=out[i]*(double)(i+1)/10000.0; printf("d=%.2f\n",d); fflush(stdout);\
  }

#define BENCH31(KERNEL,N,M,NAME,ILOOP,W,H)		\
  {\
	for(i=0;i<h*NLOOP;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<w;i++){\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/100.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*h*NLOOP, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*w, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M><<<1,32*M>>>(dout, dmat, dvec, ILOOP,W,H);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*h*NLOOP, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M><<<1,32*M>>>(dout, dmat, dvec, ILOOP,W,H);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<h*NLOOP;i++)d+=out[i]*(double)(i+1)/10000.0; printf("d=%.2f\n",d); fflush(stdout);\
  }

#define BENCH40(KERNEL,N,M,L,NAME,ILOOP)			\
  {\
	for(i=0;i<h*NLOOP;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<w;i++){\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/100.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*h*NLOOP, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*w, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M,L><<<1,32*M>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*h*NLOOP, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M,L><<<1,32*M>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<h*NLOOP;i++)d+=out[i]*(double)(i+1)/10000.0; printf("d=%.2f\n",d); fflush(stdout);\
  }

#define BENCH41(KERNEL,N,M,L,NAME,ILOOP,W,H)		\
  {\
	for(i=0;i<h*NLOOP;i++){\
	  out[i] = 0.0;\
	}\
	for(i=0;i<w;i++){\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/100.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*h*NLOOP, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*w, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M,L><<<1,32*M>>>(dout, dmat, dvec, ILOOP,W,H);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*h*NLOOP, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N,M,L><<<1,32*M>>>(dout, dmat, dvec, ILOOP,W,H);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<h*NLOOP;i++)d+=out[i]*(double)(i+1)/10000.0; printf("d=%.2f\n",d); fflush(stdout);\
  }

  // W-major
  /*
  printf("W-major 0\n");
  for(x=0;x<2;x++){
	BENCH1(gpukernelW32,  " W32", IEXEC);
	BENCH1(gpukernelW16r, "W16r", IEXEC);
	BENCH1(gpukernelW16c, "W16c", IEXEC);
	BENCH1(gpukernelW8r,  " W8r", IEXEC);
	BENCH1(gpukernelW8c,  " W8c", IEXEC);
	BENCH1(gpukernelW4r,  " W4r", IEXEC);
	BENCH1(gpukernelW4c,  " W4c", IEXEC);
	BENCH1(gpukernelW2r,  " W2r", IEXEC);
	BENCH1(gpukernelW2c,  " W2c", IEXEC);
	BENCH1(gpukernelW1,   "  W1", IEXEC);
  }
  */

  /*
for n in 32 16 8 4 2 1
do
echo "printf(\"W-major A${n}\\n\");"
for m in `seq 1 32`
do
echo "BENCH3(gpukernelAMWNr, ${n},  ${m}, \"A${m}MW${n}r\", IEXEC);"
done
done

for n in 32 16 8 4 2 1
do
echo "printf(\"W-major B${n}\\n\");"
for m in `seq 1 32`
do
echo "BENCH3(gpukernelBMWNr, ${n},  ${m}, \"B${m}MW${n}r\", IEXEC);"
done
done
   */

  BENCH30(gpukernelAMWNr, 32,  1, "A1MW32r", IEXEC);


  // 32x32 only kernel
#ifdef KERNEL32
  for(x=0;x<1;x++){
printf("W-major A32\n");
BENCH30(gpukernelAMWNr2, 32,  1, "AM01WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  2, "AM02WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  3, "AM03WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  4, "AM04WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  5, "AM05WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  6, "AM06WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  7, "AM07WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  8, "AM08WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  9, "AM09WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  10, "AM10WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  11, "AM11WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  12, "AM12WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  13, "AM13WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  14, "AM14WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  15, "AM15WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  16, "AM16WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  17, "AM17WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  18, "AM18WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  19, "AM19WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  20, "AM20WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  21, "AM21WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  22, "AM22WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  23, "AM23WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  24, "AM24WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  25, "AM25WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  26, "AM26WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  27, "AM27WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  28, "AM28WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  29, "AM29WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  30, "AM30WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  31, "AM31WN32r", IEXEC);
BENCH30(gpukernelAMWNr2, 32,  32, "AM32WN32r", IEXEC);
printf("W-major A16\n");
BENCH30(gpukernelAMWNr2, 16,  1, "AM01WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  2, "AM02WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  3, "AM03WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  4, "AM04WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  5, "AM05WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  6, "AM06WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  7, "AM07WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  8, "AM08WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  9, "AM09WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  10, "AM10WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  11, "AM11WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  12, "AM12WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  13, "AM13WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  14, "AM14WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  15, "AM15WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  16, "AM16WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  17, "AM17WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  18, "AM18WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  19, "AM19WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  20, "AM20WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  21, "AM21WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  22, "AM22WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  23, "AM23WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  24, "AM24WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  25, "AM25WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  26, "AM26WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  27, "AM27WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  28, "AM28WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  29, "AM29WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  30, "AM30WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  31, "AM31WN16r", IEXEC);
BENCH30(gpukernelAMWNr2, 16,  32, "AM32WN16r", IEXEC);
printf("W-major A8\n");
BENCH30(gpukernelAMWNr2, 8,  1, "AM01WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  2, "AM02WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  3, "AM03WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  4, "AM04WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  5, "AM05WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  6, "AM06WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  7, "AM07WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  8, "AM08WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  9, "AM09WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  10, "AM10WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  11, "AM11WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  12, "AM12WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  13, "AM13WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  14, "AM14WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  15, "AM15WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  16, "AM16WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  17, "AM17WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  18, "AM18WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  19, "AM19WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  20, "AM20WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  21, "AM21WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  22, "AM22WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  23, "AM23WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  24, "AM24WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  25, "AM25WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  26, "AM26WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  27, "AM27WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  28, "AM28WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  29, "AM29WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  30, "AM30WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  31, "AM31WN08r", IEXEC);
BENCH30(gpukernelAMWNr2, 8,  32, "AM32WN08r", IEXEC);
printf("W-major A4\n");
BENCH30(gpukernelAMWNr2, 4,  1, "AM01WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  2, "AM02WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  3, "AM03WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  4, "AM04WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  5, "AM05WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  6, "AM06WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  7, "AM07WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  8, "AM08WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  9, "AM09WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  10, "AM10WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  11, "AM11WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  12, "AM12WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  13, "AM13WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  14, "AM14WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  15, "AM15WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  16, "AM16WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  17, "AM17WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  18, "AM18WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  19, "AM19WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  20, "AM20WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  21, "AM21WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  22, "AM22WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  23, "AM23WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  24, "AM24WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  25, "AM25WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  26, "AM26WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  27, "AM27WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  28, "AM28WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  29, "AM29WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  30, "AM30WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  31, "AM31WN04r", IEXEC);
BENCH30(gpukernelAMWNr2, 4,  32, "AM32WN04r", IEXEC);
printf("W-major A2\n");
BENCH30(gpukernelAMWNr2, 2,  1, "AM01WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  2, "AM02WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  3, "AM03WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  4, "AM04WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  5, "AM05WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  6, "AM06WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  7, "AM07WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  8, "AM08WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  9, "AM09WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  10, "AM10WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  11, "AM11WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  12, "AM12WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  13, "AM13WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  14, "AM14WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  15, "AM15WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  16, "AM16WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  17, "AM17WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  18, "AM18WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  19, "AM19WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  20, "AM20WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  21, "AM21WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  22, "AM22WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  23, "AM23WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  24, "AM24WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  25, "AM25WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  26, "AM26WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  27, "AM27WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  28, "AM28WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  29, "AM29WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  30, "AM30WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  31, "AM31WN02r", IEXEC);
BENCH30(gpukernelAMWNr2, 2,  32, "AM32WN02r", IEXEC);
printf("W-major A1\n");
BENCH30(gpukernelAMWNr2, 1,  1, "AM01WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  2, "AM02WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  3, "AM03WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  4, "AM04WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  5, "AM05WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  6, "AM06WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  7, "AM07WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  8, "AM08WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  9, "AM09WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  10, "AM10WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  11, "AM11WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  12, "AM12WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  13, "AM13WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  14, "AM14WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  15, "AM15WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  16, "AM16WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  17, "AM17WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  18, "AM18WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  19, "AM19WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  20, "AM20WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  21, "AM21WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  22, "AM22WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  23, "AM23WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  24, "AM24WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  25, "AM25WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  26, "AM26WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  27, "AM27WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  28, "AM28WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  29, "AM29WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  30, "AM30WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  31, "AM31WN01r", IEXEC);
BENCH30(gpukernelAMWNr2, 1,  32, "AM32WN01r", IEXEC);
  }

  for(x=0;x<1;x++){
printf("W-major B32\n");
BENCH30(gpukernelBMWNr2, 32,  1, "BM01W32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  2, "BM02WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  3, "BM03WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  4, "BM04WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  5, "BM05WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  6, "BM06WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  7, "BM07WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  8, "BM08WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  9, "BM09WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  10, "BM10WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  11, "BM11WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  12, "BM12WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  13, "BM13WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  14, "BM14WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  15, "BM15WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  16, "BM16WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  17, "BM17WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  18, "BM18WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  19, "BM19WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  20, "BM20WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  21, "BM21WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  22, "BM22WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  23, "BM23WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  24, "BM24WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  25, "BM25WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  26, "BM26WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  27, "BM27WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  28, "BM28WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  29, "BM29WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  30, "BM30WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  31, "BM31WN32r", IEXEC);
BENCH30(gpukernelBMWNr2, 32,  32, "BM32WN32r", IEXEC);
printf("W-major B16\n");
BENCH30(gpukernelBMWNr2, 16,  1, "BM01W16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  2, "BM02WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  3, "BM03WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  4, "BM04WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  5, "BM05WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  6, "BM06WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  7, "BM07WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  8, "BM08WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  9, "BM09WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  10, "BM10WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  11, "BM11WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  12, "BM12WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  13, "BM13WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  14, "BM14WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  15, "BM15WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  16, "BM16WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  17, "BM17WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  18, "BM18WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  19, "BM19WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  20, "BM20WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  21, "BM21WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  22, "BM22WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  23, "BM23WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  24, "BM24WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  25, "BM25WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  26, "BM26WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  27, "BM27WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  28, "BM28WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  29, "BM29WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  30, "BM30WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  31, "BM31WN16r", IEXEC);
BENCH30(gpukernelBMWNr2, 16,  32, "BM32WN16r", IEXEC);
printf("W-major B8\n");
BENCH30(gpukernelBMWNr2, 8,  1, "BM01WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  2, "BM02WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  3, "BM03WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  4, "BM04WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  5, "BM05WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  6, "BM06WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  7, "BM07WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  8, "BM08WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  9, "BM09WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  10, "BM10WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  11, "BM11WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  12, "BM12WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  13, "BM13WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  14, "BM14WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  15, "BM15WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  16, "BM16WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  17, "BM17WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  18, "BM18WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  19, "BM19WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  20, "BM20WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  21, "BM21WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  22, "BM22WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  23, "BM23WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  24, "BM24WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  25, "BM25WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  26, "BM26WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  27, "BM27WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  28, "BM28WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  29, "BM29WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  30, "BM30WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  31, "BM31WN08r", IEXEC);
BENCH30(gpukernelBMWNr2, 8,  32, "BM32WN08r", IEXEC);
printf("W-major B4\n");
BENCH30(gpukernelBMWNr2, 4,  1, "BM01WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  2, "BM02WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  3, "BM03WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  4, "BM04WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  5, "BM05WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  6, "BM06WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  7, "BM07WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  8, "BM08WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  9, "BM09WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  10, "BM10WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  11, "BM11WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  12, "BM12WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  13, "BM13WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  14, "BM14WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  15, "BM15WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  16, "BM16WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  17, "BM17WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  18, "BM18WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  19, "BM19WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  20, "BM20WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  21, "BM21WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  22, "BM22WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  23, "BM23WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  24, "BM24WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  25, "BM25WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  26, "BM26WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  27, "BM27WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  28, "BM28WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  29, "BM29WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  30, "BM30WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  31, "BM31WN04r", IEXEC);
BENCH30(gpukernelBMWNr2, 4,  32, "BM32WN04r", IEXEC);
printf("W-major B2\n");
BENCH30(gpukernelBMWNr2, 2,  1, "BM01WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  2, "BM02WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  3, "BM03WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  4, "BM04WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  5, "BM05WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  6, "BM06WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  7, "BM07WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  8, "BM08WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  9, "BM09WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  10, "BM10WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  11, "BM11WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  12, "BM12WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  13, "BM13WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  14, "BM14WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  15, "BM15WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  16, "BM16WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  17, "BM17WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  18, "BM18WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  19, "BM19WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  20, "BM20WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  21, "BM21WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  22, "BM22WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  23, "BM23WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  24, "BM24WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  25, "BM25WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  26, "BM26WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  27, "BM27WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  28, "BM28WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  29, "BM29WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  30, "BM30WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  31, "BM31WN02r", IEXEC);
BENCH30(gpukernelBMWNr2, 2,  32, "BM32WN02r", IEXEC);
printf("W-major B1\n");
BENCH30(gpukernelBMWNr2, 1,  1, "BM01WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  2, "BM02WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  3, "BM03WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  4, "BM04WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  5, "BM05WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  6, "BM06WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  7, "BM07WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  8, "BM08WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  9, "BM09WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  10, "BM10WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  11, "BM11WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  12, "BM12WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  13, "BM13WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  14, "BM14WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  15, "BM15WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  16, "BM16WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  17, "BM17WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  18, "BM18WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  19, "BM19WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  20, "BM20WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  21, "BM21WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  22, "BM22WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  23, "BM23WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  24, "BM24WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  25, "BM25WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  26, "BM26WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  27, "BM27WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  28, "BM28WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  29, "BM29WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  30, "BM30WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  31, "BM31WN01r", IEXEC);
BENCH30(gpukernelBMWNr2, 1,  32, "BM32WN01r", IEXEC);
  }
#endif

  // free size kernel
#ifndef KERNEL32
  for(x=0;x<1;x++){
printf("W-major A32\n");
BENCH31(gpukernelAMWNr3, 32,  1, "AM01WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  2, "AM02WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  3, "AM03WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  4, "AM04WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  5, "AM05WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  6, "AM06WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  7, "AM07WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  8, "AM08WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  9, "AM09WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  10, "AM10WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  11, "AM11WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  12, "AM12WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  13, "AM13WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  14, "AM14WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  15, "AM15WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  16, "AM16WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  17, "AM17WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  18, "AM18WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  19, "AM19WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  20, "AM20WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  21, "AM21WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  22, "AM22WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  23, "AM23WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  24, "AM24WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  25, "AM25WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  26, "AM26WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  27, "AM27WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  28, "AM28WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  29, "AM29WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  30, "AM30WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  31, "AM31WN32r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 32,  32, "AM32WN32r", IEXEC, w, h);
printf("W-major A16\n");
BENCH31(gpukernelAMWNr3, 16,  1, "AM01WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  2, "AM02WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  3, "AM03WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  4, "AM04WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  5, "AM05WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  6, "AM06WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  7, "AM07WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  8, "AM08WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  9, "AM09WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  10, "AM10WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  11, "AM11WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  12, "AM12WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  13, "AM13WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  14, "AM14WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  15, "AM15WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  16, "AM16WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  17, "AM17WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  18, "AM18WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  19, "AM19WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  20, "AM20WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  21, "AM21WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  22, "AM22WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  23, "AM23WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  24, "AM24WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  25, "AM25WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  26, "AM26WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  27, "AM27WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  28, "AM28WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  29, "AM29WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  30, "AM30WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  31, "AM31WN16r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 16,  32, "AM32WN16r", IEXEC, w, h);
printf("W-major A8\n");
BENCH31(gpukernelAMWNr3, 8,  1, "AM01WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  2, "AM02WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  3, "AM03WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  4, "AM04WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  5, "AM05WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  6, "AM06WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  7, "AM07WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  8, "AM08WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  9, "AM09WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  10, "AM10WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  11, "AM11WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  12, "AM12WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  13, "AM13WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  14, "AM14WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  15, "AM15WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  16, "AM16WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  17, "AM17WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  18, "AM18WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  19, "AM19WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  20, "AM20WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  21, "AM21WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  22, "AM22WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  23, "AM23WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  24, "AM24WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  25, "AM25WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  26, "AM26WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  27, "AM27WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  28, "AM28WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  29, "AM29WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  30, "AM30WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  31, "AM31WN08r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 8,  32, "AM32WN08r", IEXEC, w, h);
printf("W-major A4\n");
BENCH31(gpukernelAMWNr3, 4,  1, "AM01WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  2, "AM02WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  3, "AM03WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  4, "AM04WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  5, "AM05WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  6, "AM06WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  7, "AM07WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  8, "AM08WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  9, "AM09WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  10, "AM10WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  11, "AM11WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  12, "AM12WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  13, "AM13WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  14, "AM14WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  15, "AM15WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  16, "AM16WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  17, "AM17WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  18, "AM18WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  19, "AM19WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  20, "AM20WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  21, "AM21WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  22, "AM22WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  23, "AM23WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  24, "AM24WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  25, "AM25WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  26, "AM26WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  27, "AM27WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  28, "AM28WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  29, "AM29WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  30, "AM30WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  31, "AM31WN04r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 4,  32, "AM32WN04r", IEXEC, w, h);
printf("W-major A2\n");
BENCH31(gpukernelAMWNr3, 2,  1, "AM01WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  2, "AM02WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  3, "AM03WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  4, "AM04WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  5, "AM05WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  6, "AM06WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  7, "AM07WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  8, "AM08WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  9, "AM09WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  10, "AM10WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  11, "AM11WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  12, "AM12WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  13, "AM13WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  14, "AM14WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  15, "AM15WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  16, "AM16WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  17, "AM17WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  18, "AM18WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  19, "AM19WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  20, "AM20WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  21, "AM21WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  22, "AM22WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  23, "AM23WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  24, "AM24WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  25, "AM25WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  26, "AM26WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  27, "AM27WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  28, "AM28WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  29, "AM29WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  30, "AM30WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  31, "AM31WN02r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 2,  32, "AM32WN02r", IEXEC, w, h);
printf("W-major A1\n");
BENCH31(gpukernelAMWNr3, 1,  1, "AM01WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  2, "AM02WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  3, "AM03WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  4, "AM04WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  5, "AM05WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  6, "AM06WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  7, "AM07WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  8, "AM08WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  9, "AM09WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  10, "AM10WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  11, "AM11WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  12, "AM12WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  13, "AM13WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  14, "AM14WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  15, "AM15WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  16, "AM16WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  17, "AM17WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  18, "AM18WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  19, "AM19WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  20, "AM20WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  21, "AM21WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  22, "AM22WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  23, "AM23WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  24, "AM24WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  25, "AM25WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  26, "AM26WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  27, "AM27WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  28, "AM28WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  29, "AM29WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  30, "AM30WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  31, "AM31WN01r", IEXEC, w, h);
BENCH31(gpukernelAMWNr3, 1,  32, "AM32WN01r", IEXEC, w, h);
  }

  for(x=0;x<1;x++){
printf("W-major B32\n");
BENCH31(gpukernelBMWNr3, 32,  1, "BM01W32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  2, "BM02WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  3, "BM03WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  4, "BM04WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  5, "BM05WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  6, "BM06WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  7, "BM07WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  8, "BM08WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  9, "BM09WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  10, "BM10WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  11, "BM11WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  12, "BM12WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  13, "BM13WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  14, "BM14WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  15, "BM15WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  16, "BM16WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  17, "BM17WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  18, "BM18WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  19, "BM19WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  20, "BM20WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  21, "BM21WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  22, "BM22WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  23, "BM23WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  24, "BM24WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  25, "BM25WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  26, "BM26WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  27, "BM27WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  28, "BM28WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  29, "BM29WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  30, "BM30WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  31, "BM31WN32r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 32,  32, "BM32WN32r", IEXEC, w, h);
printf("W-major B16\n");
BENCH31(gpukernelBMWNr3, 16,  1, "BM01W16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  2, "BM02WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  3, "BM03WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  4, "BM04WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  5, "BM05WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  6, "BM06WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  7, "BM07WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  8, "BM08WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  9, "BM09WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  10, "BM10WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  11, "BM11WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  12, "BM12WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  13, "BM13WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  14, "BM14WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  15, "BM15WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  16, "BM16WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  17, "BM17WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  18, "BM18WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  19, "BM19WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  20, "BM20WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  21, "BM21WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  22, "BM22WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  23, "BM23WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  24, "BM24WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  25, "BM25WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  26, "BM26WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  27, "BM27WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  28, "BM28WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  29, "BM29WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  30, "BM30WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  31, "BM31WN16r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 16,  32, "BM32WN16r", IEXEC, w, h);
printf("W-major B8\n");
BENCH31(gpukernelBMWNr3, 8,  1, "BM01WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  2, "BM02WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  3, "BM03WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  4, "BM04WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  5, "BM05WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  6, "BM06WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  7, "BM07WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  8, "BM08WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  9, "BM09WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  10, "BM10WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  11, "BM11WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  12, "BM12WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  13, "BM13WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  14, "BM14WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  15, "BM15WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  16, "BM16WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  17, "BM17WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  18, "BM18WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  19, "BM19WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  20, "BM20WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  21, "BM21WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  22, "BM22WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  23, "BM23WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  24, "BM24WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  25, "BM25WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  26, "BM26WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  27, "BM27WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  28, "BM28WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  29, "BM29WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  30, "BM30WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  31, "BM31WN08r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 8,  32, "BM32WN08r", IEXEC, w, h);
printf("W-major B4\n");
BENCH31(gpukernelBMWNr3, 4,  1, "BM01WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  2, "BM02WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  3, "BM03WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  4, "BM04WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  5, "BM05WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  6, "BM06WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  7, "BM07WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  8, "BM08WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  9, "BM09WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  10, "BM10WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  11, "BM11WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  12, "BM12WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  13, "BM13WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  14, "BM14WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  15, "BM15WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  16, "BM16WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  17, "BM17WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  18, "BM18WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  19, "BM19WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  20, "BM20WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  21, "BM21WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  22, "BM22WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  23, "BM23WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  24, "BM24WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  25, "BM25WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  26, "BM26WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  27, "BM27WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  28, "BM28WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  29, "BM29WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  30, "BM30WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  31, "BM31WN04r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 4,  32, "BM32WN04r", IEXEC, w, h);
printf("W-major B2\n");
BENCH31(gpukernelBMWNr3, 2,  1, "BM01WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  2, "BM02WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  3, "BM03WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  4, "BM04WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  5, "BM05WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  6, "BM06WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  7, "BM07WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  8, "BM08WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  9, "BM09WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  10, "BM10WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  11, "BM11WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  12, "BM12WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  13, "BM13WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  14, "BM14WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  15, "BM15WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  16, "BM16WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  17, "BM17WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  18, "BM18WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  19, "BM19WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  20, "BM20WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  21, "BM21WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  22, "BM22WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  23, "BM23WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  24, "BM24WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  25, "BM25WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  26, "BM26WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  27, "BM27WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  28, "BM28WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  29, "BM29WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  30, "BM30WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  31, "BM31WN02r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 2,  32, "BM32WN02r", IEXEC, w, h);
printf("W-major B1\n");
BENCH31(gpukernelBMWNr3, 1,  1, "BM01WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  2, "BM02WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  3, "BM03WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  4, "BM04WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  5, "BM05WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  6, "BM06WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  7, "BM07WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  8, "BM08WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  9, "BM09WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  10, "BM10WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  11, "BM11WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  12, "BM12WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  13, "BM13WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  14, "BM14WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  15, "BM15WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  16, "BM16WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  17, "BM17WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  18, "BM18WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  19, "BM19WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  20, "BM20WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  21, "BM21WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  22, "BM22WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  23, "BM23WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  24, "BM24WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  25, "BM25WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  26, "BM26WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  27, "BM27WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  28, "BM28WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  29, "BM29WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  30, "BM30WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  31, "BM31WN01r", IEXEC, w, h);
BENCH31(gpukernelBMWNr3, 1,  32, "BM32WN01r", IEXEC, w, h);
  }
#endif

  // CLMWNr
  // N M L
  // M WARPs/TB = 32*M threads / TB
  // N*L threads : 1 GEMV, N threads/line

  // BENCH40: 32x32 only kernel
  // BENCH41: free size kernel

#ifdef KERNEL32
  BENCH40(gpukernelCLMWNr2, 32,   2,  1, "CL01M02WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   2,  2, "CL02M02WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   4,  1, "CL01M04WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   4,  2, "CL02M04WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   4,  4, "CL04M04WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   8,  1, "CL01M08WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   8,  2, "CL02M08WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   8,  4, "CL04M08WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,   8,  8, "CL08M08WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  16,  1, "CL01M16WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  16,  2, "CL02M16WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  16,  4, "CL04M16WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  16,  8, "CL08M16WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  16, 16, "CL16M16WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  32,  1, "CL01M32WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  32,  2, "CL02M32WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  32,  4, "CL04M32WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  32,  8, "CL08M32WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  32, 16, "CL16M32WN32r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 32,  32, 32, "CL32M32WN32r", IEXEC);

  BENCH40(gpukernelCLMWNr2, 16,   2,  1, "CL01M02WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   2,  2, "CL02M02WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   4,  1, "CL01M04WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   4,  2, "CL02M04WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   4,  4, "CL04M04WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   8,  1, "CL01M08WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   8,  2, "CL02M08WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   8,  4, "CL04M08WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,   8,  8, "CL08M08WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  16,  1, "CL01M16WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  16,  2, "CL02M16WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  16,  4, "CL04M16WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  16,  8, "CL08M16WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  16, 16, "CL16M16WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  32,  1, "CL01M32WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  32,  2, "CL02M32WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  32,  4, "CL04M32WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  32,  8, "CL08M32WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  32, 16, "CL16M32WN16r", IEXEC);
  BENCH40(gpukernelCLMWNr2, 16,  32, 32, "CL32M32WN16r", IEXEC);

  BENCH40(gpukernelCLMWNr2,  8,   2,  1, "CL01M02WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   2,  2, "CL02M02WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   4,  1, "CL01M04WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   4,  2, "CL02M04WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   4,  4, "CL04M04WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   8,  1, "CL01M08WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   8,  2, "CL02M08WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   8,  4, "CL04M08WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,   8,  8, "CL08M08WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  16,  1, "CL01M16WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  16,  2, "CL02M16WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  16,  4, "CL04M16WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  16,  8, "CL08M16WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  16, 16, "CL16M16WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  32,  1, "CL01M32WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  32,  2, "CL02M32WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  32,  4, "CL04M32WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  32,  8, "CL08M32WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  32, 16, "CL16M32WN08r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  8,  32, 32, "CL32M32WN08r", IEXEC);

  BENCH40(gpukernelCLMWNr2,  4,   2,  1, "CL01M02WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   2,  2, "CL02M02WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   4,  1, "CL01M04WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   4,  2, "CL02M04WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   4,  4, "CL04M04WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   8,  1, "CL01M08WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   8,  2, "CL02M08WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   8,  4, "CL04M08WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,   8,  8, "CL08M08WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  16,  1, "CL01M16WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  16,  2, "CL02M16WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  16,  4, "CL04M16WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  16,  8, "CL08M16WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  16, 16, "CL16M16WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  32,  1, "CL01M32WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  32,  2, "CL02M32WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  32,  4, "CL04M32WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  32,  8, "CL08M32WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  32, 16, "CL16M32WN04r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  4,  32, 32, "CL32M32WN04r", IEXEC);

  BENCH40(gpukernelCLMWNr2,  2,   2,  1, "CL01M02WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   2,  2, "CL02M02WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   4,  1, "CL01M04WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   4,  2, "CL02M04WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   4,  4, "CL04M04WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   8,  1, "CL01M08WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   8,  2, "CL02M08WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   8,  4, "CL04M08WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,   8,  8, "CL08M08WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  16,  1, "CL01M16WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  16,  2, "CL02M16WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  16,  4, "CL04M16WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  16,  8, "CL08M16WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  16, 16, "CL16M16WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  32,  1, "CL01M32WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  32,  2, "CL02M32WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  32,  4, "CL04M32WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  32,  8, "CL08M32WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  32, 16, "CL16M32WN02r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  2,  32, 32, "CL32M32WN02r", IEXEC);

  BENCH40(gpukernelCLMWNr2,  1,   2,  1, "CL01M02WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   2,  2, "CL02M02WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   4,  1, "CL01M04WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   4,  2, "CL02M04WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   4,  4, "CL04M04WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   8,  1, "CL01M08WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   8,  2, "CL02M08WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   8,  4, "CL04M08WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,   8,  8, "CL08M08WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  16,  1, "CL01M16WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  16,  2, "CL02M16WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  16,  4, "CL04M16WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  16,  8, "CL08M16WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  16, 16, "CL16M16WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  32,  1, "CL01M32WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  32,  2, "CL02M32WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  32,  4, "CL04M32WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  32,  8, "CL08M32WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  32, 16, "CL16M32WN01r", IEXEC);
  BENCH40(gpukernelCLMWNr2,  1,  32, 32, "CL32M32WN01r", IEXEC);
#endif

#ifndef KERNEL32
  BENCH41(gpukernelCLMWNr3, 32,   2,  1, "CL01M02WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   2,  2, "CL02M02WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   4,  1, "CL01M04WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   4,  2, "CL02M04WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   4,  4, "CL04M04WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   8,  1, "CL01M08WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   8,  2, "CL02M08WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   8,  4, "CL04M08WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,   8,  8, "CL08M08WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  16,  1, "CL01M16WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  16,  2, "CL02M16WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  16,  4, "CL04M16WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  16,  8, "CL08M16WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  16, 16, "CL16M16WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  32,  1, "CL01M32WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  32,  2, "CL02M32WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  32,  4, "CL04M32WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  32,  8, "CL08M32WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  32, 16, "CL16M32WN32r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 32,  32, 32, "CL32M32WN32r", IEXEC, w, h);

  BENCH41(gpukernelCLMWNr3, 16,   2,  1, "CL01M02WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   2,  2, "CL02M02WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   4,  1, "CL01M04WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   4,  2, "CL02M04WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   4,  4, "CL04M04WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   8,  1, "CL01M08WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   8,  2, "CL02M08WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   8,  4, "CL04M08WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,   8,  8, "CL08M08WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  16,  1, "CL01M16WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  16,  2, "CL02M16WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  16,  4, "CL04M16WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  16,  8, "CL08M16WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  16, 16, "CL16M16WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  32,  1, "CL01M32WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  32,  2, "CL02M32WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  32,  4, "CL04M32WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  32,  8, "CL08M32WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  32, 16, "CL16M32WN16r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3, 16,  32, 32, "CL32M32WN16r", IEXEC, w, h);

  BENCH41(gpukernelCLMWNr3,  8,   2,  1, "CL01M02WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   2,  2, "CL02M02WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   4,  1, "CL01M04WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   4,  2, "CL02M04WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   4,  4, "CL04M04WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   8,  1, "CL01M08WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   8,  2, "CL02M08WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   8,  4, "CL04M08WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,   8,  8, "CL08M08WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  16,  1, "CL01M16WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  16,  2, "CL02M16WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  16,  4, "CL04M16WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  16,  8, "CL08M16WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  16, 16, "CL16M16WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  32,  1, "CL01M32WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  32,  2, "CL02M32WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  32,  4, "CL04M32WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  32,  8, "CL08M32WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  32, 16, "CL16M32WN08r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  8,  32, 32, "CL32M32WN08r", IEXEC, w, h);

  BENCH41(gpukernelCLMWNr3,  4,   2,  1, "CL01M02WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   2,  2, "CL02M02WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   4,  1, "CL01M04WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   4,  2, "CL02M04WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   4,  4, "CL04M04WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   8,  1, "CL01M08WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   8,  2, "CL02M08WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   8,  4, "CL04M08WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,   8,  8, "CL08M08WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  16,  1, "CL01M16WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  16,  2, "CL02M16WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  16,  4, "CL04M16WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  16,  8, "CL08M16WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  16, 16, "CL16M16WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  32,  1, "CL01M32WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  32,  2, "CL02M32WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  32,  4, "CL04M32WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  32,  8, "CL08M32WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  32, 16, "CL16M32WN04r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  4,  32, 32, "CL32M32WN04r", IEXEC, w, h);

  BENCH41(gpukernelCLMWNr3,  2,   2,  1, "CL01M02WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   2,  2, "CL02M02WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   4,  1, "CL01M04WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   4,  2, "CL02M04WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   4,  4, "CL04M04WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   8,  1, "CL01M08WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   8,  2, "CL02M08WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   8,  4, "CL04M08WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,   8,  8, "CL08M08WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  16,  1, "CL01M16WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  16,  2, "CL02M16WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  16,  4, "CL04M16WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  16,  8, "CL08M16WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  16, 16, "CL16M16WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  32,  1, "CL01M32WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  32,  2, "CL02M32WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  32,  4, "CL04M32WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  32,  8, "CL08M32WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  32, 16, "CL16M32WN02r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  2,  32, 32, "CL32M32WN02r", IEXEC, w, h);

  BENCH41(gpukernelCLMWNr3,  1,   2,  1, "CL01M02WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   2,  2, "CL02M02WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   4,  1, "CL01M04WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   4,  2, "CL02M04WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   4,  4, "CL04M04WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   8,  1, "CL01M08WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   8,  2, "CL02M08WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   8,  4, "CL04M08WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,   8,  8, "CL08M08WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  16,  1, "CL01M16WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  16,  2, "CL02M16WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  16,  4, "CL04M16WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  16,  8, "CL08M16WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  16, 16, "CL16M16WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  32,  1, "CL01M32WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  32,  2, "CL02M32WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  32,  4, "CL04M32WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  32,  8, "CL08M32WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  32, 16, "CL16M32WN01r", IEXEC, w, h);
  BENCH41(gpukernelCLMWNr3,  1,  32, 32, "CL32M32WN01r", IEXEC, w, h);
#endif

  hipFree(dout); hipFree(dmat); hipFree(dvec);
  free(out); free(mat); free(vec);
  return 0;
}
