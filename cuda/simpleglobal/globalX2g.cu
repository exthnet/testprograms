// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define NEXEC 1000

__global__ void gpukernel1(double *out, double *in, int iloop)
{
  int id;
  int y;
  double tmp = 0.0;
  for(id=0;id<NEXEC;id++){
	for(y=0; y<32; y++){
	  tmp += in[y*32000+threadIdx.x+id*32];
	}
  }
  out[threadIdx.x] += tmp;
}

__global__ void gpukernel2(double *out, double *in, int iloop)
{
  int id;
  int x;
  double tmp = 0.0;
  for(id=0;id<NEXEC;id++){
	for(x=0; x<32; x++){
	  tmp += in[threadIdx.x*32000+x+id*32];
	}
  }
  out[threadIdx.x] += tmp;
}

// ######## ######## ######## ######## ######## ######## ######## ########

int main(int argc, char **argv)
{
  int N = 1000;
  int len = N * 32 * 32;
  int i, x;
  double *out, *in;
  double *dout, *din;
  double d;

  if(argc>1)N=atoi(argv[1]); printf("N=%d\n", N);
  out = (double*)malloc(sizeof(double)*32);
  in = (double*)malloc(sizeof(double)*len);

  hipMalloc((void**)&dout, sizeof(double)*32);
  hipMalloc((void**)&din, sizeof(double)*len);

  {
	x = 1;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	}
	for(i=0;i<len;i++){
	  in[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }

  {
	x = 2;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	}
	for(i=0;i<len;i++){
	  in[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel2<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel2<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }

  {
	x = 1;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	}
	for(i=0;i<len;i++){
	  in[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel1<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }

  {
	x = 2;

	for(i=0;i<32;i++){
	  out[i] = 0.0;
	}
	for(i=0;i<len;i++){
	  in[i] = (double)(i+1);
	}

	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
	hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);

	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel2<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

	d=omp_get_wtime();
	for(i=0;i<NEXEC;i++){
	  hipDeviceSynchronize();
	  gpukernel2<<<1,32>>>(dout, din, NEXEC);
	  hipDeviceSynchronize();
	}
	d=omp_get_wtime()-d;

	printf("%2d: time %f msec(/total), ", x, d);
	d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);
  }


  hipFree(dout);  hipFree(din);
  free(out); free(in);
  return 0;
}
