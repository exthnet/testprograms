// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

__global__ void gpukernel1(double *out, double *in, int N)
{
  int i;
  int begin = (threadIdx.x/2)*(N/16)+(threadIdx.x%2);
  int end = ((threadIdx.x/2)+1)*(N/16);
  int step = 2;
  double tmp = 0.0;
  for(i=begin;i<end;i+=step){
	tmp += in[i];
  }
  out[threadIdx.x] = tmp;
}

// ######## ######## ######## ######## ######## ######## ######## ########

int main(int argc, char **argv)
{
  int N = 100;
  int len = N * 32;
  int i;
  double *out, *in;
  double *dout, *din;
  double d;

  if(argc>1)N=atoi(argv[1]); printf("N=%d, ", N);
  out = (double*)malloc(sizeof(double)*32);
  in = (double*)malloc(sizeof(double)*len);

  for(i=0;i<32;i++){
	out[i] = 0.0;
  }
  for(i=0;i<len;i++){
	in[i] = (double)(i+1);
  }

  hipMalloc((void**)&dout, sizeof(double)*32);
  hipMalloc((void**)&din, sizeof(double)*len);

  hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);
  hipMemcpy(din, in, sizeof(double)*len, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  gpukernel1<<<1,32>>>(dout, din, N);
  hipDeviceSynchronize();
  hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);

  d=omp_get_wtime();
  for(i=0;i<100;i++){
	hipDeviceSynchronize();
	gpukernel1<<<1,32>>>(dout, din, N);
	hipDeviceSynchronize();
  }
  d=omp_get_wtime()-d;

  printf("%2d: time %f msec(/100times), ", 16, d);
  d=0.0; for(i=0;i<32;i++)d+=out[i]; printf("d=%.2f\n",d);

  hipFree(dout);  hipFree(din);
  free(out); free(in);
  return 0;
}

