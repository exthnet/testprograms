// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define NLOOP 1000
#define NEXEC 10
#define IEXEC NLOOP

/*
W=32
  0  1  2  3 ... 31
W=16
  0  1  2  3 ... 15
 16 17 18 19 ... 31
W=8
  0  1  2  3 ...  7
  8  9 10 11 ... 15
 16 17 18 19 ... 23
 24 25 26 27 ... 31
W=4
  0  1  2  3
  4  5  6  7
  ...
 28 29 30 31
W=2
  0  1
  2  3
 ...
 30 31
W=1
  0
  1
  2
  3
 ...
 31
*/

__global__ void gpukernelW32(double *out, double *mat, double *vec, int iloop)
{
  int i, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y++){
	  tmp = 0.0;
	  tmp += mat[y*32 + threadIdx.x + i*32*32];// * vec[threadIdx.x];
	  if(threadIdx.x==0)out[y] += tmp;
	}
  }
}

__global__ void gpukernelW16r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=2){
	  tmp = 0.0;
	  for(x=0; x<32; x+=16){
		tmp += mat[(y+threadIdx.x/16)*32 + threadIdx.x%16+x + i*32*32];// * vec[threadIdx.x%16+x];
	  }
	  if(threadIdx.x%16==0)out[y+threadIdx.x/16] += tmp;
	}
  }
}

__global__ void gpukernelW16c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=16){
	  for(y=0; y<32; y+=2){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x/16)*32 + threadIdx.x%16+x + i*32*32];// * vec[threadIdx.x%16+x];
		if(threadIdx.x%16==0)out[y+threadIdx.x/16] += tmp;
	  }
	}
  }
}

__global__ void gpukernelW8r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=4){
	  tmp = 0.0;
	  for(x=0; x<32; x+=8){
		tmp += mat[(y+threadIdx.x/8)*32 + threadIdx.x%8+x + i*32*32];// * vec[threadIdx.x%8+x];
	  }
	  if(threadIdx.x%8==0)out[y+threadIdx.x/8] += tmp;
	}
  }
}

__global__ void gpukernelW8c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=8){
	  for(y=0; y<32; y+=4){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x/8)*32 + threadIdx.x%8+x + i*32*32];// * vec[threadIdx.x%8+x];
		if(threadIdx.x%8==0)out[y+threadIdx.x/8] += tmp;
	  }
	}
  }
}

__global__ void gpukernelW4r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=8){
	  tmp = 0.0;
	  for(x=0; x<32; x+=4){
		tmp += mat[(y+threadIdx.x/4)*32 + threadIdx.x%4+x + i*32*32];// * vec[threadIdx.x%4+x];
	  }
	  if(threadIdx.x%4==0)out[y+threadIdx.x/4] += tmp;
	}
  }
}

__global__ void gpukernelW4c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=4){
	  for(y=0; y<32; y+=8){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x/4)*32 + threadIdx.x%4+x + i*32*32];// * vec[threadIdx.x%4+x];
		if(threadIdx.x%4==0)out[y+threadIdx.x/4] += tmp;
	  }
	}
  }
}

__global__ void gpukernelW2r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=16){
	  tmp = 0.0;
	  for(x=0; x<32; x+=2){
		tmp += mat[(y+threadIdx.x/2)*32 + threadIdx.x%2+x + i*32*32];// * vec[threadIdx.x%2+x];
	  }
	  if(threadIdx.x%2==0)out[y+threadIdx.x/2] += tmp;
	}
  }
}

__global__ void gpukernelW2c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=2){
	  for(y=0; y<32; y+=16){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x/2)*32 + threadIdx.x%2+x + i*32*32];// * vec[threadIdx.x%2+x];
		if(threadIdx.x%2==0)out[y+threadIdx.x/2] += tmp;
	  }
	}
  }
}

__global__ void gpukernelW1(double *out, double *mat, double *vec, int iloop)
{
  int i, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	tmp = 0.0;
	for(x=0; x<32; x++){
	  tmp += mat[threadIdx.x*32+x + i*32*32];// * vec[x];
	}
	out[threadIdx.x] += tmp;
  }
}

// ######## ######## ######## ######## ######## ######## ######## ########

__global__ void gpukernelH32(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	tmp = 0.0;
	for(x=0; x<32; x+=1){
	  tmp += mat[threadIdx.x*32 + x + i*32*32];// * vec[x];
	}
	out[threadIdx.x] += tmp;
  }
}

__global__ void gpukernelH16r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=16){
	  tmp = 0.0;
	  for(x=0; x<32; x+=2){
		tmp += mat[(y+threadIdx.x%16)*32 + threadIdx.x/16+x + i*32*32];// * vec[threadIdx.x/16+x];
	  }
	  if(threadIdx.x/16==0)out[y+threadIdx.x] += tmp;
	}
  }
}

__global__ void gpukernelH16c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=2){
	  for(y=0; y<32; y+=16){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x%16)*32 + threadIdx.x/16+x + i*32*32];// * vec[threadIdx.x/16+x];
		if(threadIdx.x/16==0)out[y+threadIdx.x] += tmp;
	  }
	}
  }
}

__global__ void gpukernelH8r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=8){
	  tmp = 0.0;
	  for(x=0; x<32; x+=4){
		tmp += mat[(y+threadIdx.x%8)*32 + threadIdx.x/8+x + i*32*32];// * vec[threadIdx.x/8+x];
	  }
	  if(threadIdx.x/8==0)out[y+threadIdx.x] += tmp;
	}
  }
}

__global__ void gpukernelH8c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=4){
	  for(y=0; y<32; y+=8){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x%8)*32 + threadIdx.x/8+x + i*32*32];// * vec[threadIdx.x/8+x];
		if(threadIdx.x/8==0)out[y+threadIdx.x] += tmp;
	  }
	}
  }
}

__global__ void gpukernelH4r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=4){
	  tmp = 0.0;
	  for(x=0; x<32; x+=8){
		tmp += mat[(y+threadIdx.x%4)*32 + threadIdx.x/4+x + i*32*32];// * vec[threadIdx.x/4+x];
	  }
	  if(threadIdx.x/4==0)out[y+threadIdx.x] += tmp;
	}
  }
}

__global__ void gpukernelH4c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=8){
	  for(y=0; y<32; y+=4){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x%4)*32 + threadIdx.x/4+x + i*32*32];// * vec[threadIdx.x/4+x];
		if(threadIdx.x/4==0)out[y+threadIdx.x] += tmp;
	  }
	}
  }
}

__global__ void gpukernelH2r(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=2){
	  tmp = 0.0;
	  for(x=0; x<32; x+=16){
		tmp += mat[(y+threadIdx.x%2)*32 + threadIdx.x/2+x + i*32*32];// * vec[threadIdx.x/2+x];
	  }
	  if(threadIdx.x/2==0)out[y+threadIdx.x] += tmp;
	}
  }
}

__global__ void gpukernelH2c(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=16){
	  for(y=0; y<32; y+=2){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x%2)*32 + threadIdx.x/2+x + i*32*32];// * vec[threadIdx.x/2+x];
		if(threadIdx.x/2==0)out[y+threadIdx.x] += tmp;
	  }
	}
  }
}

__global__ void gpukernelH1(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=1){
	  tmp = 0.0;
	  tmp += mat[y*32 + threadIdx.x + i*32*32];// * vec[threadIdx.x];
	  if(threadIdx.x==0)out[y] += tmp;
	}
  }
}

// ######## ######## ######## ######## ######## ######## ######## ########

// general
template<int N>
__global__ void gpukernelWNr(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=32/N){
	  tmp = 0.0;
	  for(x=0; x<32; x+=N){
		tmp += mat[(y+threadIdx.x/N)*32 + threadIdx.x%N+x + i*32*32];// * vec[threadIdx.x%N+x];
	  }
	  if(threadIdx.x%N==0)out[y+threadIdx.x/N] += tmp;
	}
  }
}
template<int N>
__global__ void gpukernelWNc(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=N){
	  for(y=0; y<32; y+=32/N){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x/N)*32 + threadIdx.x%N+x + i*32*32];// * vec[threadIdx.x%N+x];
		if(threadIdx.x%N==0)out[y+threadIdx.x/N] += tmp;
	  }
	}
  }
}

template<int N>
__global__ void gpukernelWNr2(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=threadIdx.x/N; y<32; y+=32/N){
	  tmp = 0.0;
	  for(x=threadIdx.x%N; x<32; x+=N){
		tmp += mat[y*32 + x + i*32*32];// * vec[x];
	  }
	  if(threadIdx.x%N==0)out[y] += tmp;
	}
  }
}
template<int N>
__global__ void gpukernelWNc2(double *out, double *mat, double *vec, int iloop)
{
  int i, x, y;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=threadIdx.x%N; x<32; x+=N){
	  for(y=threadIdx.x/N; y<32; y+=32/N){
		tmp = 0.0;
		tmp += mat[y*32 + x + i*32*32];// * vec[x];
		if(threadIdx.x%N==0)out[y] += tmp;
	  }
	}
  }
}

// ######## ######## ######## ######## ######## ######## ######## ########

template<int N>
__global__ void gpukernelHNr(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=0; y<32; y+=N){
	  tmp = 0.0;
	  for(x=0; x<32; x+=32/N){
		tmp += mat[(y+threadIdx.x%N)*32 + threadIdx.x/N+x + i*32*32];// * vec[threadIdx.x/N+x];
	  }
	  if(threadIdx.x/N==0)out[y+threadIdx.x%N] += tmp;
	}
  }
}

template<int N>
__global__ void gpukernelHNc(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=0; x<32; x+=32/N){
	  for(y=0; y<32; y+=N){
		tmp = 0.0;
		tmp += mat[(y+threadIdx.x%N)*32 + threadIdx.x/N+x + i*32*32];// * vec[threadIdx.x/N+x];
		if(threadIdx.x/N==0)out[y+threadIdx.x%N] += tmp;
	  }
	}
  }
}

template<int N>
__global__ void gpukernelHNr2(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(y=threadIdx.x%N; y<32; y+=N){
	  tmp = 0.0;
	  for(x=threadIdx.x/N; x<32; x+=32/N){
		tmp += mat[y*32 + x + i*32*32];// * vec[x];
	  }
	  if(threadIdx.x/N==0)out[y] += tmp;
	}
  }
}

template<int N>
__global__ void gpukernelHNc2(double *out, double *mat, double *vec, int iloop)
{
  int i, y, x;
  double tmp = 0.0;
  for(i=0;i<iloop;i++){
	for(x=threadIdx.x/N; x<32; x+=32/N){
	  for(y=threadIdx.x%N; y<32; y+=N){
		tmp = 0.0;
		tmp += mat[y*32 + x + i*32*32];// * vec[x];
		if(threadIdx.x/N==0)out[y] += tmp;
	  }
	}
  }
}

// ######## ######## ######## ######## ######## ######## ######## ########

int main(int argc, char **argv)
{
  int len = NLOOP * 32 * 32;
  int i, x;
  double *out, *mat, *vec;
  double *dout, *dmat, *dvec;
  double d;

  out = (double*)malloc(sizeof(double)*32);
  mat = (double*)malloc(sizeof(double)*len);
  vec = (double*)malloc(sizeof(double)*32);

  hipMalloc((void**)&dout, sizeof(double)*32);
  hipMalloc((void**)&dmat, sizeof(double)*len);
  hipMalloc((void**)&dvec, sizeof(double)*32);

#define BENCH1(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<<<1,32>>>(dout, dmat, dvec, ILOOP);				\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<<<1,32>>>(dout, dmat, dvec, ILOOP);				\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]*(double)(i+1)/32.0; printf("d=%.2f\n",d);\
  }

#define BENCH2(KERNEL,N,NAME,ILOOP)				\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  KERNEL<N><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]*(double)(i+1)/32.0; printf("d=%.2f\n",d);\
  }

#define BENCH2r(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNr<KERNEL><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNr<KERNEL><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]*(double)(i+1)/32.0; printf("d=%.2f\n",d);\
  }

#define BENCH2c(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNc<KERNEL><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNc<KERNEL><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]*(double)(i+1)/32.0; printf("d=%.2f\n",d);\
  }

#define BENCH2r2(KERNEL,NAME,ILOOP)						\
  {\
	for(i=0;i<32;i++){\
	  out[i] = 0.0;\
	  vec[i] = sin((double)i/10.0);\
	}\
	for(i=0;i<len;i++){\
	  mat[i] = (double)(i+1)/10.0;				\
	}\
\
	hipMemcpy(dout, out, sizeof(double)*32, hipMemcpyHostToDevice);\
	hipMemcpy(dmat, mat, sizeof(double)*len, hipMemcpyHostToDevice);\
	hipMemcpy(dvec, vec, sizeof(double)*32, hipMemcpyHostToDevice);\
\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNr2<KERNEL><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	hipMemcpy(out, dout, sizeof(double)*32, hipMemcpyDeviceToHost);\
\
	d=omp_get_wtime();\
	for(i=0;i<NEXEC;i++){\
	  hipDeviceSynchronize();\
	  gpukernelWNr2<KERNEL><<<1,32>>>(dout, dmat, dvec, ILOOP);	\
	  hipDeviceSynchronize();\
	}\
	d=omp_get_wtime()-d;\
\
	printf("%s: time %f msec(/total), ", NAME, d);\
	d=0.0; for(i=0;i<32;i++)d+=out[i]*(double)(i+1)/32.0; printf("d=%.2f\n",d);	\
  }


  for(x=0;x<2;x++){
	BENCH1(gpukernelW32,  " W32", 1);
	BENCH1(gpukernelW16r, "W16r", 1);
	BENCH1(gpukernelW16c, "W16c", 1);
	BENCH1(gpukernelW8r,  " W8r", 1);
	BENCH1(gpukernelW8c,  " W8c", 1);
	BENCH1(gpukernelW4r,  " W4r", 1);
	BENCH1(gpukernelW4c,  " W4c", 1);
	BENCH1(gpukernelW2r,  " W2r", 1);
	BENCH1(gpukernelW2c,  " W2c", 1);
	BENCH1(gpukernelW1,   "  W1", 1);
  }
  for(x=0;x<2;x++){
	BENCH1(gpukernelW32,  " W32", IEXEC);
	BENCH1(gpukernelW16r, "W16r", IEXEC);
	BENCH1(gpukernelW16c, "W16c", IEXEC);
	BENCH1(gpukernelW8r,  " W8r", IEXEC);
	BENCH1(gpukernelW8c,  " W8c", IEXEC);
	BENCH1(gpukernelW4r,  " W4r", IEXEC);
	BENCH1(gpukernelW4c,  " W4c", IEXEC);
	BENCH1(gpukernelW2r,  " W2r", IEXEC);
	BENCH1(gpukernelW2c,  " W2c", IEXEC);
	BENCH1(gpukernelW1,   "  W1", IEXEC);
  }

  for(x=0;x<2;x++){
	BENCH2r(32,  "W32r", IEXEC);
	BENCH2c(32,  "W32c", IEXEC);
	BENCH2r(32,  "W32r", IEXEC);
	BENCH2c(32,  "W32c", IEXEC);
	BENCH2r(16,  "W16r", IEXEC);
	BENCH2c(16,  "W16c", IEXEC);
	BENCH2r(8,   " W8r", IEXEC);
	BENCH2c(8,   " W8c", IEXEC);
	BENCH2r(4,   " W4r", IEXEC);
	BENCH2c(4,   " W4c", IEXEC);
	BENCH2r(2,   " W2r", IEXEC);
	BENCH2c(2,   " W2c", IEXEC);
	BENCH2r(1,   " W1r", IEXEC);
	BENCH2c(1,   " W1c", IEXEC);
  }

  for(x=0;x<2;x++){
	BENCH2r2(32,  "W32r2", IEXEC);
	BENCH2r2(32,  "W32r2", IEXEC);
	BENCH2r2(16,  "W16r2", IEXEC);
	BENCH2r2(8,   " W8r2", IEXEC);
	BENCH2r2(4,   " W4r2", IEXEC);
	BENCH2r2(2,   " W2r2", IEXEC);
	BENCH2r2(1,   " W1r2", IEXEC);
  }

  for(x=0;x<2;x++){
	BENCH2(gpukernelWNr, 32, "W32r", IEXEC);
	BENCH2(gpukernelWNc, 32, "W32c", IEXEC);
	BENCH2(gpukernelWNr, 32, "W32r", IEXEC);
	BENCH2(gpukernelWNc, 32, "W32c", IEXEC);
	BENCH2(gpukernelWNr, 16, "W16r", IEXEC);
	BENCH2(gpukernelWNc, 16, "W16c", IEXEC);
	BENCH2(gpukernelWNr,  8, " W8r", IEXEC);
	BENCH2(gpukernelWNc,  8, " W8c", IEXEC);
	BENCH2(gpukernelWNr,  4, " W4r", IEXEC);
	BENCH2(gpukernelWNc,  4, " W4c", IEXEC);
	BENCH2(gpukernelWNr,  2, " W2r", IEXEC);
	BENCH2(gpukernelWNc,  2, " W2c", IEXEC);
	BENCH2(gpukernelWNr,  1, " W1r", IEXEC);
	BENCH2(gpukernelWNc,  1, " W1c", IEXEC);
  }

  // W-major
  printf("W-major 0\n");
  for(x=0;x<2;x++){
	BENCH1(gpukernelW32,  " W32", IEXEC);
	BENCH1(gpukernelW16r, "W16r", IEXEC);
	BENCH1(gpukernelW16c, "W16c", IEXEC);
	BENCH1(gpukernelW8r,  " W8r", IEXEC);
	BENCH1(gpukernelW8c,  " W8c", IEXEC);
	BENCH1(gpukernelW4r,  " W4r", IEXEC);
	BENCH1(gpukernelW4c,  " W4c", IEXEC);
	BENCH1(gpukernelW2r,  " W2r", IEXEC);
	BENCH1(gpukernelW2c,  " W2c", IEXEC);
	BENCH1(gpukernelW1,   "  W1", IEXEC);
  }

  printf("W-major 1\n");
  for(x=0;x<2;x++){
	BENCH2(gpukernelWNr, 32, "W32r", IEXEC);
	BENCH2(gpukernelWNc, 32, "W32c", IEXEC);
	BENCH2(gpukernelWNr, 16, "W16r", IEXEC);
	BENCH2(gpukernelWNc, 16, "W16c", IEXEC);
	BENCH2(gpukernelWNr,  8, " W8r", IEXEC);
	BENCH2(gpukernelWNc,  8, " W8c", IEXEC);
	BENCH2(gpukernelWNr,  4, " W4r", IEXEC);
	BENCH2(gpukernelWNc,  4, " W4c", IEXEC);
	BENCH2(gpukernelWNr,  2, " W2r", IEXEC);
	BENCH2(gpukernelWNc,  2, " W2c", IEXEC);
	BENCH2(gpukernelWNr,  1, " W1r", IEXEC);
	BENCH2(gpukernelWNc,  1, " W1c", IEXEC);
  }

  printf("W-major 2\n");
  for(x=0;x<2;x++){
	BENCH2(gpukernelWNr2, 32, "W32r2", IEXEC);
	BENCH2(gpukernelWNc2, 32, "W32c2", IEXEC);
	BENCH2(gpukernelWNr2, 16, "W16r2", IEXEC);
	BENCH2(gpukernelWNc2, 16, "W16c2", IEXEC);
	BENCH2(gpukernelWNr2,  8, " W8r2", IEXEC);
	BENCH2(gpukernelWNc2,  8, " W8c2", IEXEC);
	BENCH2(gpukernelWNr2,  4, " W4r2", IEXEC);
	BENCH2(gpukernelWNc2,  4, " W4c2", IEXEC);
	BENCH2(gpukernelWNr2,  2, " W2r2", IEXEC);
	BENCH2(gpukernelWNc2,  2, " W2c2", IEXEC);
	BENCH2(gpukernelWNr2,  1, " W1r2", IEXEC);
	BENCH2(gpukernelWNc2,  1, " W1c2", IEXEC);
  }

  // H-major
  printf("H-major 0\n");
  for(x=0;x<2;x++){
	BENCH1(gpukernelH32,  " H32", IEXEC);
	BENCH1(gpukernelH16r, "H16r", IEXEC);
	BENCH1(gpukernelH16c, "H16c", IEXEC);
	BENCH1(gpukernelH8r,  " H8r", IEXEC);
	BENCH1(gpukernelH8c,  " H8c", IEXEC);
	BENCH1(gpukernelH4r,  " H4r", IEXEC);
	BENCH1(gpukernelH4c,  " H4c", IEXEC);
	BENCH1(gpukernelH2r,  " H2r", IEXEC);
	BENCH1(gpukernelH2c,  " H2c", IEXEC);
	BENCH1(gpukernelH1,   "  H1", IEXEC);
  }

  printf("H-major 1\n");
  for(x=0;x<2;x++){
	BENCH2(gpukernelHNr, 32, "H32r", IEXEC);
	BENCH2(gpukernelHNc, 32, "H32c", IEXEC);
	BENCH2(gpukernelHNr, 16, "H16r", IEXEC);
	BENCH2(gpukernelHNc, 16, "H16c", IEXEC);
	BENCH2(gpukernelHNr,  8, " H8r", IEXEC);
	BENCH2(gpukernelHNc,  8, " H8c", IEXEC);
	BENCH2(gpukernelHNr,  4, " H4r", IEXEC);
	BENCH2(gpukernelHNc,  4, " H4c", IEXEC);
	BENCH2(gpukernelHNr,  2, " H2r", IEXEC);
	BENCH2(gpukernelHNc,  2, " H2c", IEXEC);
	BENCH2(gpukernelHNr,  1, " H1r", IEXEC);
	BENCH2(gpukernelHNc,  1, " H1c", IEXEC);
  }

  printf("H-major 2\n");
  for(x=0;x<2;x++){
	BENCH2(gpukernelHNr2, 32, "H32r2", IEXEC);
	BENCH2(gpukernelHNc2, 32, "H32c2", IEXEC);
	BENCH2(gpukernelHNr2, 16, "H16r2", IEXEC);
	BENCH2(gpukernelHNc2, 16, "H16c2", IEXEC);
	BENCH2(gpukernelHNr2,  8, " H8r2", IEXEC);
	BENCH2(gpukernelHNc2,  8, " H8c2", IEXEC);
	BENCH2(gpukernelHNr2,  4, " H4r2", IEXEC);
	BENCH2(gpukernelHNc2,  4, " H4c2", IEXEC);
	BENCH2(gpukernelHNr2,  2, " H2r2", IEXEC);
	BENCH2(gpukernelHNc2,  2, " H2c2", IEXEC);
	BENCH2(gpukernelHNr2,  1, " H1r2", IEXEC);
	BENCH2(gpukernelHNc2,  1, " H1c2", IEXEC);
  }

  hipFree(dout); hipFree(dmat); hipFree(dvec);
  free(out); free(mat); free(vec);
  return 0;
}
