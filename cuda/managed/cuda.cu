// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
//#include <cuda_rt.h>

__global__ void gpukernel(int N, double *C, double *A, double *B)
{
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  C[id] = A[id] * B[id];
}


int main(int argc, char **argv)
{
  int i, j, N;
  double *A, *B, *C;
  double *dA, *dB, *dC;

  N = 4;
  A = (double*)malloc(sizeof(double)*N*N);
  B = (double*)malloc(sizeof(double)*N*N);
  C = (double*)malloc(sizeof(double)*N*N);

  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  C[i*N+j] = 0.0f;
	  B[i*N+j] = 2.0f;
	  A[i*N+j] = (double)(i)/(double)(N)*1000.0 + (double)(j)/(double)(N);
	}
  }

  hipMalloc((void**)&dA, sizeof(double)*N*N);
  hipMalloc((void**)&dB, sizeof(double)*N*N);
  hipMalloc((void**)&dC, sizeof(double)*N*N);

  printf("A\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", A[i*N+j]);
	}
	printf("\n");
  }
  printf("B\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", B[i*N+j]);
	}
	printf("\n");
  }
  printf("C (before)\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", C[i*N+j]);
	}
	printf("\n");
  }

  hipMemcpy(dA, A, sizeof(double)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(double)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, sizeof(double)*N*N, hipMemcpyHostToDevice);

  dim3 grids;
  dim3 blocks;
  grids = dim3(4, 1, 1);
  blocks = dim3(4, 1 ,1);
  gpukernel<<<grids,blocks>>>(N, dC, dA, dB);

  hipMemcpy(C, dC, sizeof(double)*N*N, hipMemcpyDeviceToHost);

  printf("C (after)\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", C[i*N+j]);
	}
	printf("\n");
  }

  hipFree(dA); hipFree(dB); hipFree(dC);
  free(A); free(B); free(C);
  return 0;
}
