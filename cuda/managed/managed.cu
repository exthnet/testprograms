// -*- c++ -*-
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
//#include <cuda_rt.h>

__global__ void gpukernel(int N, double *C, double *A, double *B)
{
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  C[id] = A[id] * B[id];
}


int main(int argc, char **argv)
{
  int i, j, N;
  double *A, *B, *C;

  N = 4;
  hipMallocManaged((void**)&A, sizeof(double)*N*N);
  hipMallocManaged((void**)&B, sizeof(double)*N*N);
  hipMallocManaged((void**)&C, sizeof(double)*N*N);

  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  C[i*N+j] = 0.0f;
	  B[i*N+j] = 2.0f;
	  A[i*N+j] = (double)(i)/(double)(N)*1000.0 + (double)(j)/(double)(N);
	}
  }

  printf("A\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", A[i*N+j]);
	}
	printf("\n");
  }
  printf("B\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", B[i*N+j]);
	}
	printf("\n");
  }
  printf("C (before)\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", C[i*N+j]);
	}
	printf("\n");
  }

  dim3 grids;
  dim3 blocks;
  grids = dim3(4, 1, 1);
  blocks = dim3(4, 1 ,1);
  gpukernel<<<grids,blocks>>>(N, C, A, B);
  hipDeviceSynchronize();

  printf("C (after)\n");
  for(i=0; i<N; i++){
	for(j=0; j<N; j++){
	  printf(" %2.4f", C[i*N+j]);
	}
	printf("\n");
  }

  hipFree(A); hipFree(B); hipFree(C);
  return 0;
}
