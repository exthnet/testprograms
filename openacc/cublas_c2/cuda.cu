// -*- c++ -*-

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

__global__ void cudakernel(int N, float *C, float *A, float *B)
{
  int i, j, k;
  i = blockIdx.x;
  j = threadIdx.x;
  for(k=0; k<N; k++){
	C[i*N+j] += A[i*N+k] * B[k*N+j];
  }
}

int main(int argc, char **argv)
{
  int i, j;
  int N;
  float *A, *B, *C;
  float *dA, *dB, *dC;
  struct timeval tv1, tv2;

  if(argc!=2){
	printf("usage: %s N\n", argv[0]);
	return -1;
  }

  N = atoi(argv[1]);
  printf("N = %d\n", N);

  A = (float*)malloc(sizeof(float)*N*N);
  B = (float*)malloc(sizeof(float)*N*N);
  C = (float*)malloc(sizeof(float)*N*N);
  hipMalloc((void**)&dA, sizeof(float)*N);
  hipMalloc((void**)&dB, sizeof(float)*N);
  hipMalloc((void**)&dC, sizeof(float)*N);

  for(i=0;i<N;i++){
	for(j=0;j<N;j++){
	  C[i*N+j] = 0.0f;
	  A[i*N+j] = (float)(i) + (float)j/10.0f;
	  B[i*N+j] = (float)(i) + (float)j/10.0f;
	}
  }

  hipMemcpy(dA, A, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, sizeof(float)*N, hipMemcpyHostToDevice);
  dim3 grids;
  dim3 blocks;
  grids = dim3(2, 1, 1);
  blocks = dim3(64, 1 ,1);
  gettimeofday(&tv1,NULL);
  cudakernel<<<grids,blocks>>>(N, dC, dA, dB);
  gettimeofday(&tv2,NULL);
  hipMemcpy(C, dC, sizeof(float)*N, hipMemcpyDeviceToHost);

  printf("TIME: %d %f\n", N, (tv2.tv_sec+tv2.tv_usec*1e-6)-(tv1.tv_sec+tv1.tv_usec*1e-6));
  hipFree(dA); hipFree(dB); hipFree(dC);
  free(A); free(B); free(C);
  return 0;
}
