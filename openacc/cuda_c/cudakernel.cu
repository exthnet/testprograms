
#include <hip/hip_runtime.h>
// -*- c++ -*-
__global__ void gpukernel(int N, float *C, float *A, float *B)
{
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  C[id] += A[id] * B[id];
}

extern "C" void gpukernel_wrapper(int N, float *C, float *A, float *B)
{
  dim3 grids;
  dim3 blocks;
  grids = dim3(2, 1, 1);
  blocks = dim3(64, 1 ,1);
  gpukernel<<<grids,blocks>>>(N, C, A, B);
}
