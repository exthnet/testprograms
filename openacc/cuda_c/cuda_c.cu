// -*- c++ -*-

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
//#include <cuda.h>

__global__ void gpukernel(int N, float *C, float *A, float *B)
{
  int id = blockIdx.x*blockDim.x + threadIdx.x;
  if(id<N)C[id] += A[id] * B[id];
}


int main(int argc, char **argv)
{
  int i, N, x;
  float *A, *B, *C;
  float *dA, *dB, *dC;

  x = 10;
  N = 128;
  A = (float*)malloc(sizeof(float)*N);
  B = (float*)malloc(sizeof(float)*N);
  C = (float*)malloc(sizeof(float)*N);

  for(i=0;i<N;i++){
	C[i] = 0.0f;	B[i] = 2.0f;
	A[i] = (float)(i+1)/(float)(N);
  }

  hipMalloc((void**)&dA, sizeof(float)*N);
  hipMalloc((void**)&dB, sizeof(float)*N);
  hipMalloc((void**)&dC, sizeof(float)*N);

  printf("A\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.4f\n", A[i]);
	}else{
	  printf(" %2.4f", A[i]);
	}
  }
  printf("\n");
  printf("B\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.4f\n", B[i]);
	}else{
	  printf(" %2.4f", B[i]);
	}
  }
  printf("\n");
  printf("C (before)\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.4f\n", C[i]);
	}else{
	  printf(" %2.4f", C[i]);
	}
  }
  printf("\n");

  hipMemcpy(dA, A, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, sizeof(float)*N, hipMemcpyHostToDevice);

  dim3 grids;
  dim3 blocks;
  grids = dim3(4, 1, 1);
  blocks = dim3(64, 1 ,1);
  gpukernel<<<grids,blocks>>>(N, dC, dA, dB);

  hipMemcpy(C, dC, sizeof(float)*N, hipMemcpyDeviceToHost);

  printf("C (after)\n");
  for(i=0; i<N; i++){
	if(i%x==x-1){
	  printf(" %2.4f\n", C[i]);
	}else{
	  printf(" %2.4f", C[i]);
	}
  }
  printf("\n");

  hipFree(dA); hipFree(dB); hipFree(dC);
  free(A); free(B); free(C);
  return 0;
}
