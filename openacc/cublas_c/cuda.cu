// -*- c++ -*-

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void cudakernel(int N, float *C, float *A, float *B)
{
  int i, j, k;
  i = blockIdx.x;
  j = threadIdx.x;
  for(k=0; k<N; k++){
	C[i*N+j] += A[i*N+k] * B[k*N+j];
  }
}

int main(int argc, char **argv)
{
  int i, j;
  int N;
  float *A, *B, *C;
  float *dA, *dB, *dC;

  N = 8;
  A = (float*)malloc(sizeof(float)*N*N);
  B = (float*)malloc(sizeof(float)*N*N);
  C = (float*)malloc(sizeof(float)*N*N);
  hipMalloc((void**)&dA, sizeof(float)*N);
  hipMalloc((void**)&dB, sizeof(float)*N);
  hipMalloc((void**)&dC, sizeof(float)*N);

  for(i=0;i<N;i++){
	for(j=0;j<N;j++){
	  C[i*N+j] = 0.0f;
	  A[i*N+j] = (float)(i) + (float)j/10.0f;
	  B[i*N+j] = (float)(i) + (float)j/10.0f;
	}
  }

  printf("A\n");
  for(i=0;i<N;i++){
	for(j=0;j<N;j++){
	  printf(" %8.2f", A[i*N+j]);
	}
	printf("\n");
  }
  printf("B\n");
  for(i=0;i<N;i++) {
	for(j=0;j<N;j++){
	  printf(" %8.2f", B[i*N+j]);
	}
	printf("\n");
  }
  printf("C (before)\n");
  for(i=0;i<N;i++){
	for(j=0;j<N;j++){
	  printf(" %8.2f", C[i*N+j]);
	}
	printf("\n");
  }

  hipMemcpy(dA, A, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float)*N, hipMemcpyHostToDevice);
  hipMemcpy(dC, C, sizeof(float)*N, hipMemcpyHostToDevice);
  dim3 grids;
  dim3 blocks;
  grids = dim3(2, 1, 1);
  blocks = dim3(64, 1 ,1);
  cudakernel<<<grids,blocks>>>(N, dC, dA, dB);
  hipMemcpy(C, dC, sizeof(float)*N, hipMemcpyDeviceToHost);

  printf("C (after)\n");
  for(i=0;i<N;i++){
	for(j=0;j<N;j++){
	  printf(" %8.2f", C[i*N+j]);
	}
	printf("\n");
  }

  hipFree(dA); hipFree(dB); hipFree(dC);
  free(A); free(B); free(C);
  return 0;
}
